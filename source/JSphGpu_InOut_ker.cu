#include "hip/hip_runtime.h"
//HEAD_DSPH
/*
 <DUALSPHYSICS>  Copyright (c) 2016, Dr Jose M. Dominguez et al. (see http://dual.sphysics.org/index.php/developers/). 

 EPHYSLAB Environmental Physics Laboratory, Universidade de Vigo, Ourense, Spain.
 School of Mechanical, Aerospace and Civil Engineering, University of Manchester, Manchester, U.K.

 This file is part of DualSPHysics. 

 DualSPHysics is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by
 the Free Software Foundation, either version 3 of the License, or (at your option) any later version. 

 DualSPHysics is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details. 

 You should have received a copy of the GNU General Public License, along with DualSPHysics. If not, see <http://www.gnu.org/licenses/>. 
*/

/// \file JSphGpu_InOut_ker.cu \brief Implements functions and CUDA kernels for InOut feature.

#include "JSphGpu_InOut_ker.h"
#include <cfloat>
#include <hip/hip_math_constants.h>

namespace cusphinout{

#include "FunctionsMath_ker.cu"

//##############################################################################
//# Kernels for inlet/outlet (JSphInOut).
//# Kernels para inlet/outlet (JSphInOut).
//##############################################################################

//------------------------------------------------------------------------------
/// Mark special fluid particles to ignore.
/// Marca las particulas fluidas especiales para ignorar.
//------------------------------------------------------------------------------
__global__ void KerInOutIgnoreFluidDef(unsigned n,typecode cod,typecode codnew,typecode *code)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    if(code[p]==cod)code[p]=codnew;
  }
}

//==============================================================================
/// Mark special fluid particles to ignore.
/// Marca las particulas fluidas especiales para ignorar.
//==============================================================================
void InOutIgnoreFluidDef(unsigned n,typecode cod,typecode codnew,typecode *code){
  if(n){
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    KerInOutIgnoreFluidDef <<<sgrid,SPHBSIZE>>> (n,cod,codnew,code);
  }
}


//------------------------------------------------------------------------------
/// Returns original position of periodic particle.
//------------------------------------------------------------------------------
__device__ double3 KerInteraction_PosNoPeriodic(double3 posp1)
{
  if(CTE.periactive&1){//-xperi
    if(posp1.x<CTE.maprealposminx)                 { posp1.x-=CTE.xperincx; posp1.y-=CTE.xperincy; posp1.z-=CTE.xperincz; }
    if(posp1.x>CTE.maprealposminx+CTE.maprealsizex){ posp1.x+=CTE.xperincx; posp1.y+=CTE.xperincy; posp1.z+=CTE.xperincz; }
  }
  if(CTE.periactive&2){//-yperi
    if(posp1.y<CTE.maprealposminy)                 { posp1.x-=CTE.yperincx; posp1.y-=CTE.yperincy; posp1.z-=CTE.yperincz; }
    if(posp1.y>CTE.maprealposminy+CTE.maprealsizey){ posp1.x+=CTE.yperincx; posp1.y+=CTE.yperincy; posp1.z+=CTE.yperincz; }
  }
  if(CTE.periactive&4){//-zperi
    if(posp1.z<CTE.maprealposminz)                 { posp1.x-=CTE.zperincx; posp1.y-=CTE.zperincy; posp1.z-=CTE.zperincz; }
    if(posp1.z>CTE.maprealposminz+CTE.maprealsizez){ posp1.x+=CTE.zperincx; posp1.y+=CTE.zperincy; posp1.z+=CTE.zperincz; }
  }
  return(posp1);
}

//------------------------------------------------------------------------------
/// Updates fluid particle position according to current position.
/// Actualizacion de posicion de particulas fluidas segun posicion actual.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerUpdatePosFluid(unsigned n,unsigned pini
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  unsigned pp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pp<n){
    unsigned p=pp+pini;
    const typecode rcode=code[p];
    const bool outrhop=(CODE_GetSpecialValue(rcode)==CODE_OUTRHOP);
    cusph::KerUpdatePos<periactive>(posxy[p],posz[p],0,0,0,outrhop,p,posxy,posz,dcell,code);
  }
}

//==============================================================================
/// Updates fluid particle position according to current position.
/// Actualizacion de posicion de particulas fluidas segun posicion actual.
//==============================================================================
void UpdatePosFluid(byte periactive,unsigned n,unsigned pini
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  if(n){
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    if(periactive)KerUpdatePosFluid<true>  <<<sgrid,SPHBSIZE>>> (n,pini,posxy,posz,dcell,code);
    else          KerUpdatePosFluid<false> <<<sgrid,SPHBSIZE>>> (n,pini,posxy,posz,dcell,code);
  }
}


//------------------------------------------------------------------------------
/// Create list of fluid particles in inlet/outlet zones and updates code[].
/// Crea lista de particulas fluind en zonas inlet/outlet y actualiza code[].
//------------------------------------------------------------------------------
__global__ void KerInOutCreateList(unsigned n,unsigned pini
  ,byte convertfluidmask,byte nzone,const byte *cfgzone,const float4 *planes
  ,float3 freemin,float3 freemax
  ,const float2 *boxlimit,const double2 *posxy,const double *posz
  ,typecode *code,unsigned *listp)
{
  extern __shared__ unsigned slist[];
  //float *splanes=(float*)(slist+(n+1));
  if(!threadIdx.x)slist[0]=0;
  __syncthreads();
  const unsigned pp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pp<n){
    const unsigned p=pp+pini;
    const typecode rcode=code[p];
    if(CODE_IsNormal(rcode) || CODE_IsPeriodic(rcode)){//-It includes normal and periodic particles.
      bool select=CODE_IsFluidInout(rcode);//-Particulas ya marcadas como in/out.
      if(!select){//-Particulas no marcadas como in/out.
        const double2 rxy=posxy[p];
        const double rz=posz[p];
        if(rxy.x<=freemin.x || rxy.y<=freemin.y || rz<=freemin.z || rxy.x>=freemax.x || rxy.y>=freemax.y || rz>=freemax.z){
          byte zone=255;
          if(boxlimit!=NULL){
            for(byte cz=0;cz<nzone && zone==255;cz++)if((cfgzone[cz]&convertfluidmask)!=0){
              const float2 xlim=boxlimit[cz];
              const float2 ylim=boxlimit[nzone+cz];
              const float2 zlim=boxlimit[nzone*2+cz];
              if(xlim.x<=rxy.x && rxy.x<=xlim.y && ylim.x<=rxy.y && rxy.y<=ylim.y && zlim.x<=rz && rz<=zlim.y){
                const float4 rpla=planes[cz];
                if((rpla.x*rxy.x+rpla.y*rxy.y+rpla.z*rz+rpla.w)<0)zone=byte(cz);
              }
            }
          }
          else{
            for(byte cz=0;cz<nzone && zone==255;cz++)if((cfgzone[cz]&convertfluidmask)!=0){
              const float4 rpla=planes[cz];
              if((rpla.x*rxy.x+rpla.y*rxy.y+rpla.z*rz+rpla.w)<0)zone=byte(cz);
            }        
          }
          if(zone!=255){
            code[p]=CODE_ToFluidInout(rcode,zone);
            select=true;
          }
        }
      }
      if(select)slist[atomicAdd(slist,1)+1]=p; //-Add particle in the list.
    }
  }
  __syncthreads();
  const unsigned ns=slist[0];
  __syncthreads();
  if(!threadIdx.x && ns)slist[0]=atomicAdd((listp+n),ns);
  __syncthreads();
  if(threadIdx.x<ns){
    const unsigned cp=slist[0]+threadIdx.x;
    listp[cp]=slist[threadIdx.x+1];
  }
}

//==============================================================================
/// Create list of fluid particles in inlet/outlet zones and updates code[].
/// With stable activated reorders perioc list.
///
/// Con stable activado reordena lista de periodicas.
//==============================================================================
unsigned InOutCreateList(bool stable,unsigned n,unsigned pini
  ,byte convertfluidmask,byte nzone,const byte *cfgzone,const float4 *planes
  ,tfloat3 freemin,tfloat3 freemax
  ,const float2 *boxlimit,const double2 *posxy,const double *posz
  ,typecode *code,unsigned *listp)
{
  unsigned count=0;
  if(n){
    //-listp size list initialized to zero.
    //-Inicializa tama�o de lista listp a cero.
    hipMemset(listp+n,0,sizeof(unsigned));
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    const unsigned smem=(SPHBSIZE+1)*sizeof(unsigned); //-All fluid particles can be in in/out area and one position for counter.
    KerInOutCreateList <<<sgrid,SPHBSIZE,smem>>> (n,pini,convertfluidmask,nzone,cfgzone,planes,Float3(freemin),Float3(freemax),boxlimit,posxy,posz,code,listp);
    hipMemcpy(&count,listp+n,sizeof(unsigned),hipMemcpyDeviceToHost);
    //-Reorders list when stable has been activated.
    //-Reordena lista cuando stable esta activado.
    //if(stable && count){ //-Does not affect results.
    //  thrust::device_ptr<unsigned> dev_list(listp);
    //  thrust::sort(dev_list,dev_list+count);
    //}
  }
  return(count);
}


//------------------------------------------------------------------------------
/// Returns velocity according profile configuration (JSphInOutZone::TpVelProfile).
//------------------------------------------------------------------------------
__device__ float KerInOutCalcVel(byte vprof,const float4 &vdata,float posz){
  float vel=0;
  if(vprof==0)vel=vdata.x;  //-PVEL_Constant
  else if(vprof==1){        //-PVEL_Linear
    const float m=vdata.x;
    const float b=vdata.y;
    vel=m*posz+b;
  }
  else if(vprof==2){        //-PVEL_Parabolic
    const float a=vdata.x;
    const float b=vdata.y;
    const float c=vdata.z;
    vel=a*posz*posz+b*posz+c;
  }
  return(vel);
}

//------------------------------------------------------------------------------
/// Updates velocity and rhop of inlet/outlet particles when it is not extrapolated. 
/// Actualiza velocidad y densidad de particulas inlet/outlet cuando no es extrapolada.
//------------------------------------------------------------------------------
__global__ void KerInOutUpdateData(unsigned n,const unsigned *inoutpart
  ,byte izone,byte rmode,byte vmode,byte vprof
  ,float timestep,float zsurf,float4 veldata,float4 veldata2,float3 dirdata
  ,float coefhydro,float rhopzero,float gamma
  ,const typecode *code,const double *posz,float4 *velrhop)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const unsigned p=inoutpart[cp];
    if(izone==byte(CODE_GetIzoneFluidInout(code[p]))){
      const double rposz=posz[p];
      float4 rvelrhop=velrhop[p];
      //-Compute rhop value.
      if(rmode==0)rvelrhop.w=rhopzero; //-MRHOP_Constant
      if(rmode==1){                    //-MRHOP_Hydrostatic
        const float depth=float(double(zsurf)-rposz);
        const float rh=1.f+coefhydro*depth;     //rh=1.+rhop0*(-gravity.z)*(Dp*ptdata.GetDepth(p))/vCteB;
        rvelrhop.w=rhopzero*pow(rh,1.f/gamma);  //rhop[id]=rhop0*pow(rh,(1./gamma));
      }
      //-Compute velocity value.
      float vel=0;
      if(vmode==0){      //-MVEL_Fixed
        vel=KerInOutCalcVel(vprof,veldata,float(rposz));
      }
      else if(vmode==1){ //-MVEL_Variable
        const float vel1=KerInOutCalcVel(vprof,veldata,float(rposz));
        const float vel2=KerInOutCalcVel(vprof,veldata2,float(rposz));
        const float time1=veldata.w;
        const float time2=veldata2.w;
        if(timestep<=time1 || time1==time2)vel=vel1;
        else if(timestep>=time2)vel=vel2;
        else vel=(timestep-time1)/(time2-time1)*(vel2-vel1)+vel1;
      }
      if(vmode!=2){      //-MVEL_Extrapolated
        rvelrhop.x=vel*dirdata.x;
        rvelrhop.y=vel*dirdata.y;
        rvelrhop.z=vel*dirdata.z;
      }
      velrhop[p]=rvelrhop;
    }
  }
}

//==============================================================================
/// Updates velocity and rhop of inlet/outlet particles when it is not extrapolated. 
/// Actualiza velocidad y densidad de particulas inlet/outlet cuando no es extrapolada.
//==============================================================================
void InOutUpdateData(unsigned n,const unsigned *inoutpart
  ,byte izone,byte rmode,byte vmode,byte vprof
  ,float timestep,float zsurf,tfloat4 veldata,tfloat4 veldata2,tfloat3 dirdata
  ,float coefhydro,float rhopzero,float gamma
  ,const typecode *code,const double *posz,float4 *velrhop)
{
  if(n){
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    KerInOutUpdateData <<<sgrid,SPHBSIZE>>> (n,inoutpart,izone,rmode,vmode,vprof
      ,timestep,zsurf,Float4(veldata),Float4(veldata2),Float3(dirdata),coefhydro,rhopzero,gamma,code,posz,velrhop);
  }
}


//------------------------------------------------------------------------------
/// Updates velocity and rhop of inlet/outlet particles when it is not extrapolated. 
/// Actualiza velocidad y densidad de particulas inlet/outlet cuando no es extrapolada.
//------------------------------------------------------------------------------
__global__ void KerInoutClearInteractionVars(unsigned n,const int *inoutpart
  ,float3 *ace,float *ar,float *viscdt,float3 *shiftpos)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const unsigned p=inoutpart[cp];
    ace[p]=make_float3(0,0,0);
    ar[p]=0;
    viscdt[p]=0;
    if(shiftpos!=NULL)shiftpos[p]=make_float3(0,0,0);
  }
}

//==============================================================================
/// Updates velocity and rhop of inlet/outlet particles when it is not extrapolated. 
/// Actualiza velocidad y densidad de particulas inlet/outlet cuando no es extrapolada.
//==============================================================================
void InoutClearInteractionVars(unsigned n,const int *inoutpart
  ,float3 *ace,float *ar,float *viscdt,float3 *shiftpos)
{
  if(n){
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    KerInoutClearInteractionVars <<<sgrid,SPHBSIZE>>> (n,inoutpart,ace,ar,viscdt,shiftpos);
  }
}


//------------------------------------------------------------------------------
/// Updates velocity and rhop for M1 variable when Verlet is used. 
/// Actualiza velocidad y densidad de varible M1 cuando se usa Verlet.
//------------------------------------------------------------------------------
__global__ void KerInOutUpdateVelrhopM1(unsigned n,const int *inoutpart
  ,const float4 *velrhop,float4 *velrhopm1)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const unsigned p=inoutpart[cp];
    velrhopm1[p]=velrhop[p];
  }
}

//==============================================================================
/// Updates velocity and rhop for M1 variable when Verlet is used. 
/// Actualiza velocidad y densidad de varible M1 cuando se usa Verlet.
//==============================================================================
void InOutUpdateVelrhopM1(unsigned n,const int *inoutpart
  ,const float4 *velrhop,float4 *velrhopm1)
{
  if(n){
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    KerInOutUpdateVelrhopM1 <<<sgrid,SPHBSIZE>>> (n,inoutpart,velrhop,velrhopm1);
  }
}


//------------------------------------------------------------------------------
/// Checks particle position.
/// If particle is moved to fluid zone then it changes to fluid particle and 
/// it creates a new in/out particle.
/// If particle is moved out the domain then it changes to ignore particle.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerInOutComputeStep(unsigned n,int *inoutpart
  ,double dt,const float4 *planes,const float *width,const float4 *velrhop
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const int p=inoutpart[cp];
    //-Checks if particle was moved to fluid domain.
    const typecode rcode=code[p];
    const byte izone=byte(CODE_GetIzoneFluidInout(rcode));
    const double2 rposxy=posxy[p];
    const float displane=-cumath::KerDistPlaneSign(planes[izone],float(rposxy.x),float(rposxy.y),float(posz[p]));
    if(displane<0)inoutpart[cp]=-int(p);//-Particle is moved to fluid domain.  //-It is not necessary on GPU code.
    else if(displane>width[izone]){//-Particle is moved out in/out zone.
      code[p]=CODE_SetOutIgnore(rcode);
      inoutpart[cp]=INT_MAX;  //-It is not necessary on GPU code.
    }
  }
}

//==============================================================================
/// Checks particle position.
/// If particle is moved to fluid zone then it changes to fluid particle and 
/// it creates a new in/out particle.
/// If particle is moved out the domain then it changes to ignore particle.
//==============================================================================
void InOutComputeStep(byte periactive,unsigned n,int *inoutpart
  ,double dt,const float4 *planes,const float *width,const float4 *velrhop
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  if(n){
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    if(periactive)KerInOutComputeStep<true>  <<<sgrid,SPHBSIZE>>> (n,inoutpart,dt,planes,width,velrhop,posxy,posz,dcell,code);
    else          KerInOutComputeStep<false> <<<sgrid,SPHBSIZE>>> (n,inoutpart,dt,planes,width,velrhop,posxy,posz,dcell,code);
  }
}


//------------------------------------------------------------------------------
/// Create list for new inlet particles to create.
/// Crea lista de nuevas particulas inlet a crear.
//------------------------------------------------------------------------------
__global__ void KerInOutListCreate(unsigned n,unsigned nmax,int *inoutpart)
{
  extern __shared__ unsigned slist[];
  if(!threadIdx.x)slist[0]=0;
  __syncthreads();
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n && inoutpart[cp]<0){
    slist[atomicAdd(slist,1)+1]=unsigned(-inoutpart[cp]); //-Add particle in the list.
  }
  __syncthreads();
  const unsigned ns=slist[0];
  __syncthreads();
  if(!threadIdx.x && ns)slist[0]=n + atomicAdd((inoutpart+nmax),ns);
  __syncthreads();
  if(threadIdx.x<ns){
    const unsigned cp2=slist[0]+threadIdx.x;
    if(cp2<nmax)inoutpart[cp2]=slist[threadIdx.x+1];
  }
}

//==============================================================================
/// Create list for new inlet particles to create at end of inoutpart[]. 
/// Returns number of new particles to create.
/// 
/// Crea lista de nuevas particulas inlet a crear al final de inoutpart[].
/// Devuelve el numero de las nuevas particulas para crear.
//==============================================================================
unsigned InOutListCreate(bool stable,unsigned n,unsigned nmax,int *inoutpart)
{
  unsigned count=0;
  if(n){
    //-inoutpart size list initialized to zero.
    //-Inicializa tama�o de lista inoutpart a cero.
    hipMemset(inoutpart+nmax,0,sizeof(unsigned));
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    const unsigned smem=(SPHBSIZE+1)*sizeof(unsigned); //-All fluid particles can be in in/out area and one position for counter.
    KerInOutListCreate <<<sgrid,SPHBSIZE,smem>>> (n,nmax,inoutpart);
    hipMemcpy(&count,inoutpart+nmax,sizeof(unsigned),hipMemcpyDeviceToHost);
    //-Reorders list if it is valid and stable has been activated.
    //-Reordena lista si es valida y stable esta activado.
    if(stable && count && count<=nmax){
      thrust::device_ptr<unsigned> dev_list((unsigned*)inoutpart);
      thrust::sort(dev_list+n,dev_list+n+count);
    }
  }
  return(count);
}


//------------------------------------------------------------------------------
/// Creates new inlet particles to replace the particles moved to fluid domain.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerInOutCreateNewInlet(unsigned newn,const unsigned *newinoutpart
  ,unsigned np,unsigned idnext,typecode codenewpart,const float3 *dirdata,const float *width
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code,unsigned *idp,float4 *velrhop)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<newn){
    const int p=newinoutpart[cp];
    const byte izone=byte(CODE_GetIzoneFluidInout(code[p]));
    code[p]=codenewpart;//-Particle changes to fluid particle.
    const double dis=width[izone];
    const float3 rdirdata=dirdata[izone];
    double2 rposxy=posxy[p];
    double rposz=posz[p];
    rposxy.x-=dis*rdirdata.x;
    rposxy.y-=dis*rdirdata.y;
    rposz-=dis*rdirdata.z;
    const unsigned p2=np+cp;
    code[p2]=CODE_ToFluidInout(codenewpart,izone);
    cusph::KerUpdatePos<periactive>(rposxy,rposz,0,0,0,false,p2,posxy,posz,dcell,code);
    idp[p2]=idnext+cp;
    velrhop[p2]=make_float4(0,0,0,0);
  }
}

//==============================================================================
/// Creates new inlet particles to replace the particles moved to fluid domain.
//==============================================================================
void InOutCreateNewInlet(byte periactive,unsigned newn,const unsigned *newinoutpart
  ,unsigned np,unsigned idnext,typecode codenewpart,const float3 *dirdata,const float *width
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code,unsigned *idp,float4 *velrhop)
{
  if(newn){
    dim3 sgrid=cusph::GetGridSize(newn,SPHBSIZE);
    if(periactive)KerInOutCreateNewInlet<true>  <<<sgrid,SPHBSIZE>>> (newn,newinoutpart,np,idnext,codenewpart,dirdata,width,posxy,posz,dcell,code,idp,velrhop);
    else          KerInOutCreateNewInlet<false> <<<sgrid,SPHBSIZE>>> (newn,newinoutpart,np,idnext,codenewpart,dirdata,width,posxy,posz,dcell,code,idp,velrhop);
  }
}


//------------------------------------------------------------------------------
/// Move in/out particles according its velocity.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerInOutFillMove(unsigned n,const unsigned *inoutpart
  ,double dt,const float4 *velrhop
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const unsigned p=inoutpart[cp];
    //-Updates position of particles.
    const float4 rvelrhop=velrhop[p];
    const double dx=double(rvelrhop.x)*dt;
    const double dy=double(rvelrhop.y)*dt;
    const double dz=double(rvelrhop.z)*dt;
    cusph::KerUpdatePos<periactive>(posxy[p],posz[p],dx,dy,dz,false,p,posxy,posz,dcell,code);
  }
}

//==============================================================================
/// Move particles in/out according its velocity.
//==============================================================================
void InOutFillMove(byte periactive,unsigned n,const unsigned *inoutpart
  ,double dt,const float4 *velrhop
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  if(n){
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    if(periactive)KerInOutFillMove<true>  <<<sgrid,SPHBSIZE>>> (n,inoutpart,dt,velrhop,posxy,posz,dcell,code);
    else          KerInOutFillMove<false> <<<sgrid,SPHBSIZE>>> (n,inoutpart,dt,velrhop,posxy,posz,dcell,code);
  }
}


//------------------------------------------------------------------------------
/// Computes projection data to filling mode.
//------------------------------------------------------------------------------
__global__ void KerInOutFillProjection(unsigned n,const unsigned *inoutpart
  ,typecode codenewpart,const float4 *planes,const float *width
  ,const double2 *posxy,const double *posz
  ,typecode *code,float *prodist,double2 *proposxy,double *proposz)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const unsigned p=inoutpart[cp];
    //-Checks if particle was moved to fluid domain.
    const typecode rcode=code[p];
    const byte izone=byte(CODE_GetIzoneFluidInout(rcode));
    const double2 rposxy=posxy[p];
    const double rposz=posz[p];
    const float4 rplanes=planes[izone];
    //-Compute distance to plane.
    const double v1=(rposxy.x*rplanes.x + rposxy.y*rplanes.y + rposz*rplanes.z + rplanes.w);
    const float v2=rplanes.x*rplanes.x+rplanes.y*rplanes.y+rplanes.z*rplanes.z;
    const float displane=-float(v1/sqrt(v2));//-Equivalent to fgeo::PlaneDistSign().
    //-Calculates point on plane and distance.
    float rprodis=0;
    double rpropx=0,rpropy=0,rpropz=0;
    if(displane<0 || displane>width[izone]){
      code[p]=(displane<0? codenewpart: CODE_SetOutIgnore(rcode));
      //-if (displane<0) Particle changes to fluid particle.
      //-if (displane>Width[izone]) Particle is moved out in/out zone.
    }
    else{
      rprodis=displane; //=fabs(displane); No hace falta porque siempre es positivo cuando ok=true.
      //-Equivalent to fmath::PtOrthogonal().
      const double t=-v1/v2;
      rpropx=rposxy.x+t*rplanes.x;
      rpropy=rposxy.y+t*rplanes.y;
      rpropz=rposz+t*rplanes.z;
    }
    //-Saves results on GPU memory.
    prodist[cp]=rprodis;
    proposxy[cp]=make_double2(rpropx,rpropy);
    proposz[cp] =rpropz;
  }
}

//==============================================================================
/// Computes projection data to filling mode.
//==============================================================================
void InOutFillProjection(unsigned n,const unsigned *inoutpart
  ,typecode codenewpart,const float4 *planes,const float *width
  ,const double2 *posxy,const double *posz
  ,typecode *code,float *prodist,double2 *proposxy,double *proposz)
{
  if(n){
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    KerInOutFillProjection <<<sgrid,SPHBSIZE>>> (n,inoutpart,codenewpart,planes,width,posxy,posz,code,prodist,proposxy,proposz);
  }
}


//------------------------------------------------------------------------------
/// Removes particles above the Zsurf limit.
//------------------------------------------------------------------------------
__global__ void KerInOutRemoveZsurf(unsigned n,const unsigned *inoutpart
  ,typecode codezone,float zsurf,const double *posz
  ,typecode *code,float *prodist,double2 *proposxy,double *proposz)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const unsigned p=inoutpart[cp];
    if(code[p]==codezone && posz[p]>zsurf){
      code[p]=CODE_SetOutIgnore(code[p]);
      prodist[cp]=0;
      proposxy[cp]=make_double2(0,0);
      proposz[cp]=0;
    }
  }
}

//==============================================================================
/// Removes particles above the Zsurf limit.
//==============================================================================
void InOutRemoveZsurf(unsigned n,const unsigned *inoutpart
  ,typecode codezone,float zsurf,const double *posz
  ,typecode *code,float *prodist,double2 *proposxy,double *proposz)
{
  if(n){
    dim3 sgrid=cusph::GetGridSize(n,SPHBSIZE);
    KerInOutRemoveZsurf <<<sgrid,SPHBSIZE>>> (n,inoutpart,codezone,zsurf,posz,code,prodist,proposxy,proposz);
  }
}


//------------------------------------------------------------------------------
/// Compute maximum distance to create points in each PtPos.
/// Create list of selected ptpoints and its distance for new inlet/outlet particles.
//------------------------------------------------------------------------------
__global__ void KerInOutFillListCreate(unsigned npt
  ,const double2 *ptposxy,const double *ptposz
  ,const byte *ptzone,const float *zsurf,const float *width
  ,unsigned npropt,const float *prodist,const double2 *proposxy,const double *proposz
  ,float dpmin,float dpmin2,float dp,float *ptdist,unsigned nmax,unsigned *inoutpart)
{
  extern __shared__ unsigned slist[];
  //float *sdist=(float*)(slist+(blockDim.x+1));
  if(!threadIdx.x)slist[0]=0;
  __syncthreads();
  const unsigned cpt=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cpt<npt){
    const double2 rptxy=ptposxy[cpt];
    const double rptz=ptposz[cpt];
    float distmax=FLT_MAX;
    if(zsurf==NULL || float(rptz)<=zsurf[ptzone[cpt]]){
      distmax=0;
      for(int cpro=0;cpro<npropt;cpro++){
        const double2 propsxy=proposxy[cpro];
        const float disx=rptxy.x-propsxy.x;
        const float disy=rptxy.y-propsxy.y;
        const float disz=rptz   -proposz [cpro];
        if(disx<=dpmin && disy<=dpmin && disz<=dpmin){//-particle near to ptpoint (approx.)
          const float dist2=(disx*disx+disy*disy+disz*disz);
          if(dist2<dpmin2){//-particle near to ptpoint.
            const float dmax=prodist[cpro]+sqrt(dpmin2-dist2);
            distmax=max(distmax,dmax);
          }
        }
      }
    }
    distmax=(distmax==0? dp: distmax);
    //-Creates list of new inlet/outlet particles.
    if(distmax<width[ptzone[cpt]]){
      slist[atomicAdd(slist,1)+1]=cpt; //-Add ptpoint in the list.
      ptdist[cpt]=distmax;             //-Saves distance of ptpoint.
    }
  }
  __syncthreads();
  const unsigned ns=slist[0];
  __syncthreads();
  if(!threadIdx.x && ns)slist[0]=atomicAdd((inoutpart+nmax),ns);
  __syncthreads();
  if(threadIdx.x<ns){
    const unsigned cp2=slist[0]+threadIdx.x;
    if(cp2<nmax)inoutpart[cp2]=slist[threadIdx.x+1];
  }
}

//==============================================================================
/// Compute maximum distance to create points in each PtPos.
/// Create list of selected ptpoints and its distance for new inlet/outlet particles.
/// Returns number of new particles to create.
//==============================================================================
unsigned InOutFillListCreate(bool stable,unsigned npt
  ,const double2 *ptposxy,const double *ptposz
  ,const byte *ptzone,const float *zsurf,const float *width
  ,unsigned npropt,const float *prodist,const double2 *proposxy,const double *proposz
  ,float dpmin,float dpmin2,float dp,float *ptdist,unsigned nmax,unsigned *inoutpart)
{
  unsigned count=0;
  if(npt){
    //-inoutpart size list initialized to zero.
    //-Inicializa tama�o de lista inoutpart a cero.
    hipMemset(inoutpart+nmax,0,sizeof(unsigned));
    dim3 sgrid=cusph::GetGridSize(npt,SPHBSIZE);
    const unsigned smem=(SPHBSIZE+1)*sizeof(unsigned); //-All fluid particles can be in in/out area and one position for counter.
    KerInOutFillListCreate <<<sgrid,SPHBSIZE,smem>>> (npt,ptposxy,ptposz,ptzone,zsurf,width,npropt,prodist,proposxy,proposz,dpmin,dpmin2,dp,ptdist,nmax,inoutpart);
    hipMemcpy(&count,inoutpart+nmax,sizeof(unsigned),hipMemcpyDeviceToHost);
    //-Reorders list if it is valid and stable has been activated.
    //-Reordena lista si es valida y stable esta activado.
    if(stable && count && count<=nmax){
      thrust::device_ptr<unsigned> dev_list((unsigned*)inoutpart);
      thrust::sort(dev_list,dev_list+count);
    }
  }
  return(count);
}


//------------------------------------------------------------------------------
/// Creates new inlet/outlet particles to fill inlet/outlet domain.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerInOutFillCreate(unsigned newn,const unsigned *newinoutpart
  ,const double2 *ptposxy,const double *ptposz,const byte *ptzone,const float *ptauxdist
  ,unsigned np,unsigned idnext,typecode codenewpart,const float3 *dirdata
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code,unsigned *idp,float4 *velrhop)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<newn){
    const unsigned cpt=newinoutpart[cp];
    const byte izone=ptzone[cpt];
    const double dis=ptauxdist[cpt];
    const float3 rdirdata=dirdata[izone];
    double2 rposxy=ptposxy[cpt];
    double rposz=ptposz[cpt];
    rposxy.x-=dis*rdirdata.x;
    rposxy.y-=dis*rdirdata.y;
    rposz   -=dis*rdirdata.z;
    const unsigned p=np+cp;
    code[p]=CODE_ToFluidInout(codenewpart,izone);
    cusph::KerUpdatePos<periactive>(rposxy,rposz,0,0,0,false,p,posxy,posz,dcell,code);
    idp[p]=idnext+cp;
    velrhop[p]=make_float4(0,0,0,0);
  }
}

//==============================================================================
/// Creates new inlet/outlet particles to fill inlet/outlet domain.
//==============================================================================
void InOutFillCreate(byte periactive,unsigned newn,const unsigned *newinoutpart
  ,const double2 *ptposxy,const double *ptposz,const byte *ptzone,const float *ptauxdist
  ,unsigned np,unsigned idnext,typecode codenewpart,const float3 *dirdata
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code,unsigned *idp,float4 *velrhop)
{
  if(newn){
    dim3 sgrid=cusph::GetGridSize(newn,SPHBSIZE);
    if(periactive)KerInOutFillCreate<true>  <<<sgrid,SPHBSIZE>>> (newn,newinoutpart,ptposxy,ptposz,ptzone,ptauxdist,np,idnext,codenewpart,dirdata,posxy,posz,dcell,code,idp,velrhop);
    else          KerInOutFillCreate<false> <<<sgrid,SPHBSIZE>>> (newn,newinoutpart,ptposxy,ptposz,ptzone,ptauxdist,np,idnext,codenewpart,dirdata,posxy,posz,dcell,code,idp,velrhop);
  }
}


//------------------------------------------------------------------------------
/// Calculates maximum zsurf in fluid domain.
//------------------------------------------------------------------------------
template<unsigned blockSize> __global__ void KerInOutComputeZsurf
  (unsigned nptz,const float3 *ptzpos,float maxdist2,float zbottom
  ,int hdiv,int4 nc,unsigned cellfluid,const int2 *begincell,int3 cellzero
  ,const double2 *posxy,const double *posz,const typecode *code,float *res)
{
  extern __shared__ float sfdat[];
  const unsigned tid=threadIdx.x;

  const unsigned p1=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p1<nptz){
    float zsurfmax=zbottom;
    //-Obtains basic data of particle p1.
    const float3 posp1=ptzpos[p1];
    
    //-Obtains interaction limits.
    int cxini,cxfin,yini,yfin,zini,zfin;
    cusph::KerGetInteractionCells(posp1.x,posp1.y,posp1.z,hdiv,nc,cellzero,cxini,cxfin,yini,yfin,zini,zfin);

    //-Interaction with fluids.
    for(int z=zini;z<zfin;z++){
      int zmod=(nc.w)*z+cellfluid; //-The sum showing where fluid cells start. | Le suma donde empiezan las celdas de fluido.
      for(int y=yini;y<yfin;y++){
        int ymod=zmod+nc.x*y;
        unsigned pini,pfin=0;
        for(int x=cxini;x<cxfin;x++){
          int2 cbeg=begincell[x+ymod];
          if(cbeg.y){
            if(!pfin)pini=cbeg.x;
            pfin=cbeg.y;
          }
        }
        if(pfin)for(int p2=pini;p2<pfin;p2++){
          const float poszp2=float(posz[p2]);
          if(poszp2>zsurfmax){
            const float drz=posp1.z-poszp2;
            const double2 posxyp2=posxy[p2];
            const float drx=posp1.x-float(posxyp2.x);
            const float dry=posp1.y-float(posxyp2.y);
            const float rr2=drx*drx+dry*dry+drz*drz;
            if(rr2<=maxdist2 && CODE_IsFluidNotInout(code[p2]))zsurfmax=poszp2;//-Only with fluid particles but not inout particles.
          }
        }
      }
    }
    sfdat[tid]=zsurfmax;
  }
  else sfdat[tid]=zbottom;
  //-Reduces maximum in shared memory.
  __syncthreads();
  if(blockSize>=512){ if(tid<256)sfdat[tid]=max(sfdat[tid],sfdat[tid+256]);  __syncthreads(); }
  if(blockSize>=256){ if(tid<128)sfdat[tid]=max(sfdat[tid],sfdat[tid+128]);  __syncthreads(); }
  if(blockSize>=128){ if(tid<64) sfdat[tid]=max(sfdat[tid],sfdat[tid+64] );  __syncthreads(); }
  if(tid<32)cusph::KerReduMaxFloatWarp<blockSize>(sfdat,tid);
  if(tid==0)res[blockIdx.y*gridDim.x + blockIdx.x]=sfdat[0];
}

//==============================================================================
/// Calculates maximum zsurf in fluid domain.
//==============================================================================
float InOutComputeZsurf(unsigned nptz,const float3 *ptzpos,float maxdist,float zbottom
  ,TpCellMode cellmode,tuint3 ncells,const int2 *begincell,tuint3 cellmin
  ,const double2 *posxy,const double *posz,const typecode *code
  ,float *auxg,float *auxh)
{
  const int hdiv=(cellmode==CELLMODE_H? 2: 1);
  const int4 nc=make_int4(int(ncells.x),int(ncells.y),int(ncells.z),int(ncells.x*ncells.y));
  const unsigned cellfluid=nc.w*nc.z+1;
  const int3 cellzero=make_int3(cellmin.x,cellmin.y,cellmin.z);
  float zsurfmax=zbottom;
  if(nptz){
    const unsigned bsize=256;
    dim3 sgrid=cusph::GetGridSize(nptz,bsize);
    unsigned smem=sizeof(float)*bsize;
    KerInOutComputeZsurf<bsize> <<<sgrid,bsize,smem>>> (nptz,ptzpos,(maxdist*maxdist),zbottom,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,auxg);
    const unsigned nblocks=sgrid.x*sgrid.y;
    hipMemcpy(auxh,auxg,sizeof(float)*nblocks,hipMemcpyDeviceToHost);
    for(unsigned c=0;c<nblocks;c++)zsurfmax=max(zsurfmax,auxh[c]);
  }
  return(zsurfmax);
}


//------------------------------------------------------------------------------
/// Perform interaction between ghost inlet/outlet nodes and fluid particles. GhostNodes-Fluid
/// Realiza interaccion entre ghost inlet/outlet nodes y particulas de fluido. GhostNodes-Fluid
//------------------------------------------------------------------------------
template<bool sim2d,TpKernel tker> __global__ void KerInteractionInOutExtrap_Double
  (unsigned inoutcount,const int *inoutpart,const byte *cfgzone,byte computerhopmask,byte computevelmask
  ,const float4 *planes,const float* width,const float3 *dirdata,float determlimit
  ,int hdiv,int4 nc,unsigned cellfluid,const int2 *begincell,int3 cellzero
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<inoutcount){
    const unsigned p1=inoutpart[cp];
    const byte izone=byte(CODE_GetIzoneFluidInout(code[p1]));
    const byte cfg=cfgzone[izone];
    const bool computerhop=((cfg&computerhopmask)!=0);
    const bool computevel= ((cfg&computevelmask )!=0);
    if(computerhop || computevel){
      //-Calculates ghost node position.
      double3 pos_p1=make_double3(posxy[p1].x,posxy[p1].y,posz[p1]);
      if(CODE_IsPeriodic(code[p1]))pos_p1=KerInteraction_PosNoPeriodic(pos_p1);
      const double displane=cumath::DistPlane(planes[izone],pos_p1)*2;
      const float3 rdirdata=dirdata[izone];
      const double3 posp1=make_double3(pos_p1.x+displane*rdirdata.x, pos_p1.y+displane*rdirdata.y, pos_p1.z+displane*rdirdata.z); //-Ghost node position.

      //-Initializes variables for calculation.
      double rhopp1=0;
      double3 gradrhopp1=make_double3(0,0,0);
      double3 velp1=make_double3(0,0,0);
      tmatrix3d gradvelp1; cumath::Tmatrix3dReset(gradvelp1); //-Only for velocity.
      tmatrix3d a_corr2; if(sim2d) cumath::Tmatrix3dReset(a_corr2); //-Only for 2D.
      tmatrix4d a_corr3; if(!sim2d)cumath::Tmatrix4dReset(a_corr3); //-Only for 3D.

      //-Obtains interaction limits.
      int cxini,cxfin,yini,yfin,zini,zfin;
      cusph::KerGetInteractionCells(posp1.x,posp1.y,posp1.z,hdiv,nc,cellzero,cxini,cxfin,yini,yfin,zini,zfin);

      //-Interaction with fluids.
      for(int z=zini;z<zfin;z++){
        int zmod=(nc.w)*z+cellfluid; //-The sum showing where fluid cells start. | Le suma donde empiezan las celdas de fluido.
        for(int y=yini;y<yfin;y++){
          int ymod=zmod+nc.x*y;
          unsigned pini,pfin=0;
          for(int x=cxini;x<cxfin;x++){
            int2 cbeg=begincell[x+ymod];
            if(cbeg.y){
              if(!pfin)pini=cbeg.x;
              pfin=cbeg.y;
            }
          }
          if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
            const double2 p2xy=posxy[p2];
            const double drx=double(posp1.x-p2xy.x);
            const double dry=double(posp1.y-p2xy.y);
            const double drz=double(posp1.z-posz[p2]);
            const double rr2=drx*drx+dry*dry+drz*drz;
            if(rr2<=CTE.fourh2 && rr2>=ALMOSTZERO && CODE_IsFluidNotInout(code[p2])){//-Only with fluid particles but not inout particles.
              //-Wendland or Cubic Spline kernel.
			  float ffrx,ffry,ffrz,fwab;
			  if(tker==KERNEL_Wendland)cusph::KerGetKernelWendland(float(rr2),float(drx),float(dry),float(drz),ffrx,ffry,ffrz,fwab);
			  else if(tker==KERNEL_Cubic)cusph::KerGetKernelCubic(float(rr2),float(drx),float(dry),float(drz),ffrx,ffry,ffrz,fwab);
  		      const double frx=ffrx,fry=ffry,frz=ffrz,wab=fwab;

              const float4 velrhopp2=velrhop[p2];
              //===== Get mass and volume of particle p2 =====
              double massp2=CTE.massf;
              double volp2=massp2/velrhopp2.w;

              //===== Density and its gradient =====
              rhopp1+=massp2*wab;
              gradrhopp1.x+=massp2*frx;
              gradrhopp1.y+=massp2*fry;
              gradrhopp1.z+=massp2*frz;

              //===== Kernel values multiplied by volume =====
              const double vwab=wab*volp2;
              const double vfrx=frx*volp2;
              const double vfry=fry*volp2;
              const double vfrz=frz*volp2;

              //===== Velocity and its gradient =====
              if(computevel){
                velp1.x+=vwab*velrhopp2.x;
                velp1.y+=vwab*velrhopp2.y;
                velp1.z+=vwab*velrhopp2.z;
                gradvelp1.a11+=vfrx*velrhopp2.x;	// du/dx
                gradvelp1.a12+=vfry*velrhopp2.x;	// du/dy
                gradvelp1.a13+=vfrz*velrhopp2.x;	// du/dz
                gradvelp1.a21+=vfrx*velrhopp2.y;	// dv/dx
                gradvelp1.a22+=vfry*velrhopp2.y;	// dv/dx
                gradvelp1.a23+=vfrz*velrhopp2.y;	// dv/dx
                gradvelp1.a31+=vfrx*velrhopp2.z;	// dw/dx
                gradvelp1.a32+=vfry*velrhopp2.z;	// dw/dx
                gradvelp1.a33+=vfrz*velrhopp2.z;	// dw/dx
              }

              //===== Matrix A for correction =====
              if(sim2d){
                a_corr2.a11+=vwab; 	a_corr2.a12+=drx*vwab;	a_corr2.a13+=drz*vwab;
                a_corr2.a21+=vfrx; 	a_corr2.a22+=drx*vfrx; 	a_corr2.a23+=drz*vfrx;
                a_corr2.a31+=vfrz; 	a_corr2.a32+=drx*vfrz;	a_corr2.a33+=drz*vfrz;
              }
              else{
                a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
                a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
                a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
                a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
              }
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      float4 velrhopfinal=velrhop[p1];
      const double3 dpos=make_double3(pos_p1.x-posp1.x, pos_p1.y-posp1.y, pos_p1.z-posp1.z); //-Inlet/outlet particle position - ghost node position.
      if(sim2d){
        const double determ=cumath::Determinant3x3(a_corr2);
        if(determ>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
          const tmatrix3d invacorr2=cumath::InverseMatrix3x3(a_corr2,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const double rhoghost=rhopp1*invacorr2.a11 + gradrhopp1.x*invacorr2.a12 + gradrhopp1.z*invacorr2.a13;
            const double grx=-(rhopp1*invacorr2.a21 + gradrhopp1.x*invacorr2.a22 + gradrhopp1.z*invacorr2.a23);
            const double grz=-(rhopp1*invacorr2.a31 + gradrhopp1.x*invacorr2.a32 + gradrhopp1.z*invacorr2.a33);
            velrhopfinal.w=float(rhoghost + grx*dpos.x + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const double velghost_x=velp1.x*invacorr2.a11 + gradvelp1.a11*invacorr2.a12 + gradvelp1.a13*invacorr2.a13;
            const double velghost_z=velp1.z*invacorr2.a11 + gradvelp1.a31*invacorr2.a12 + gradvelp1.a33*invacorr2.a13;
            const double a11=-(velp1.x*invacorr2.a21 + gradvelp1.a11*invacorr2.a22 + gradvelp1.a13*invacorr2.a23);
            const double a13=-(velp1.z*invacorr2.a21 + gradvelp1.a31*invacorr2.a22 + gradvelp1.a33*invacorr2.a23);
            const double a31=-(velp1.x*invacorr2.a31 + gradvelp1.a11*invacorr2.a32 + gradvelp1.a13*invacorr2.a33);
            const double a33=-(velp1.z*invacorr2.a31 + gradvelp1.a31*invacorr2.a32 + gradvelp1.a33*invacorr2.a33);
    	    velrhopfinal.x=float(velghost_x + a11*dpos.x + a31*dpos.z);
    	    velrhopfinal.z=float(velghost_z + a13*dpos.x + a33*dpos.z);
            velrhopfinal.y=0;
   	      }
        }
        else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr2.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr2.a11);
            velrhopfinal.z=float(velp1.z/a_corr2.a11);
            velrhopfinal.y=0;
   	      }
        }
      }
      else{
        const double determ=cumath::Determinant4x4(a_corr3);
        if(determ>=determlimit){
          const tmatrix4d invacorr3=cumath::InverseMatrix4x4(a_corr3,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const double rhoghost=rhopp1*invacorr3.a11 + gradrhopp1.x*invacorr3.a12 + gradrhopp1.y*invacorr3.a13 + gradrhopp1.z*invacorr3.a14;
            const double grx=   -(rhopp1*invacorr3.a21 + gradrhopp1.x*invacorr3.a22 + gradrhopp1.y*invacorr3.a23 + gradrhopp1.z*invacorr3.a24);
            const double gry=   -(rhopp1*invacorr3.a31 + gradrhopp1.x*invacorr3.a32 + gradrhopp1.y*invacorr3.a33 + gradrhopp1.z*invacorr3.a34);
            const double grz=   -(rhopp1*invacorr3.a41 + gradrhopp1.x*invacorr3.a42 + gradrhopp1.y*invacorr3.a43 + gradrhopp1.z*invacorr3.a44);
            velrhopfinal.w=float(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const double velghost_x=velp1.x*invacorr3.a11 + gradvelp1.a11*invacorr3.a12 + gradvelp1.a12*invacorr3.a13 + gradvelp1.a13*invacorr3.a14;
      	    const double velghost_y=velp1.y*invacorr3.a11 + gradvelp1.a11*invacorr3.a12 + gradvelp1.a12*invacorr3.a13 + gradvelp1.a13*invacorr3.a14;
      	    const double velghost_z=velp1.z*invacorr3.a11 + gradvelp1.a31*invacorr3.a12 + gradvelp1.a32*invacorr3.a13 + gradvelp1.a33*invacorr3.a14;
            const double a11=-(velp1.x*invacorr3.a21 + gradvelp1.a11*invacorr3.a22 + gradvelp1.a12*invacorr3.a23 + gradvelp1.a13*invacorr3.a24);
        	const double a12=-(velp1.y*invacorr3.a21 + gradvelp1.a21*invacorr3.a22 + gradvelp1.a22*invacorr3.a23 + gradvelp1.a23*invacorr3.a24);
        	const double a13=-(velp1.z*invacorr3.a21 + gradvelp1.a31*invacorr3.a22 + gradvelp1.a32*invacorr3.a23 + gradvelp1.a33*invacorr3.a24);
        	const double a21=-(velp1.x*invacorr3.a31 + gradvelp1.a11*invacorr3.a32 + gradvelp1.a12*invacorr3.a33 + gradvelp1.a13*invacorr3.a34);
        	const double a22=-(velp1.y*invacorr3.a31 + gradvelp1.a21*invacorr3.a32 + gradvelp1.a22*invacorr3.a33 + gradvelp1.a23*invacorr3.a34);
        	const double a23=-(velp1.z*invacorr3.a31 + gradvelp1.a31*invacorr3.a32 + gradvelp1.a32*invacorr3.a33 + gradvelp1.a33*invacorr3.a34);
        	const double a31=-(velp1.x*invacorr3.a41 + gradvelp1.a11*invacorr3.a42 + gradvelp1.a12*invacorr3.a43 + gradvelp1.a13*invacorr3.a44);
        	const double a32=-(velp1.y*invacorr3.a41 + gradvelp1.a21*invacorr3.a42 + gradvelp1.a22*invacorr3.a43 + gradvelp1.a23*invacorr3.a44);
        	const double a33=-(velp1.z*invacorr3.a41 + gradvelp1.a31*invacorr3.a42 + gradvelp1.a32*invacorr3.a43 + gradvelp1.a33*invacorr3.a44);
            velrhopfinal.x=float(velghost_x + a11*dpos.x + a21*dpos.y + a31*dpos.z);
            velrhopfinal.y=float(velghost_y + a12*dpos.x + a22*dpos.y + a32*dpos.z);
      	    velrhopfinal.z=float(velghost_z + a13*dpos.x + a23*dpos.y + a33*dpos.z);
          }
        }
        else if(a_corr3.a11>0){ // Determinant is small but a11 is nonzero, 0th order ANGELO
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr3.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr3.a11);
            velrhopfinal.y=float(velp1.y/a_corr3.a11);
            velrhopfinal.z=float(velp1.z/a_corr3.a11);
     	  }
        }
      }
      velrhop[p1]=velrhopfinal;
    }
  }
}

//------------------------------------------------------------------------------
/// Perform interaction between ghost inlet/outlet nodes and fluid particles. GhostNodes-Fluid
/// Realiza interaccion entre ghost inlet/outlet nodes y particulas de fluido. GhostNodes-Fluid
//------------------------------------------------------------------------------
template<bool sim2d,TpKernel tker> __global__ void KerInteractionInOutExtrap_Single
  (unsigned inoutcount,const int *inoutpart,const byte *cfgzone,byte computerhopmask,byte computevelmask
  ,const float4 *planes,const float* width,const float3 *dirdata,float determlimit
  ,int hdiv,int4 nc,unsigned cellfluid,const int2 *begincell,int3 cellzero
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<inoutcount){
    const unsigned p1=inoutpart[cp];
    const byte izone=byte(CODE_GetIzoneFluidInout(code[p1]));
    const byte cfg=cfgzone[izone];
    const bool computerhop=((cfg&computerhopmask)!=0);
    const bool computevel= ((cfg&computevelmask )!=0);
    if(computerhop || computevel){
      //-Calculates ghost node position.
      double3 pos_p1=make_double3(posxy[p1].x,posxy[p1].y,posz[p1]);
      if(CODE_IsPeriodic(code[p1]))pos_p1=KerInteraction_PosNoPeriodic(pos_p1);
      const double displane=cumath::DistPlane(planes[izone],pos_p1)*2;
      const float3 rdirdata=dirdata[izone];
      const double3 posp1=make_double3(pos_p1.x+displane*rdirdata.x, pos_p1.y+displane*rdirdata.y, pos_p1.z+displane*rdirdata.z); //-Ghost node position.

      //-Initializes variables for calculation.
      float rhopp1=0;
      float3 gradrhopp1=make_float3(0,0,0);
      float3 velp1=make_float3(0,0,0);
      tmatrix3f gradvelp1; cumath::Tmatrix3fReset(gradvelp1); //-Only for velocity.
      tmatrix3d a_corr2; if(sim2d) cumath::Tmatrix3dReset(a_corr2); //-Only for 2D.
      tmatrix4d a_corr3; if(!sim2d)cumath::Tmatrix4dReset(a_corr3); //-Only for 3D.

      //-Obtains interaction limits.
      int cxini,cxfin,yini,yfin,zini,zfin;
      cusph::KerGetInteractionCells(posp1.x,posp1.y,posp1.z,hdiv,nc,cellzero,cxini,cxfin,yini,yfin,zini,zfin);

      //-Interaction with fluids.
      for(int z=zini;z<zfin;z++){
        int zmod=(nc.w)*z+cellfluid; //-The sum showing where fluid cells start. | Le suma donde empiezan las celdas de fluido.
        for(int y=yini;y<yfin;y++){
          int ymod=zmod+nc.x*y;
          unsigned pini,pfin=0;
          for(int x=cxini;x<cxfin;x++){
            int2 cbeg=begincell[x+ymod];
            if(cbeg.y){
              if(!pfin)pini=cbeg.x;
              pfin=cbeg.y;
            }
          }
          if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
            const double2 p2xy=posxy[p2];
            const float drx=float(posp1.x-p2xy.x);
            const float dry=float(posp1.y-p2xy.y);
            const float drz=float(posp1.z-posz[p2]);
            const float rr2=drx*drx+dry*dry+drz*drz;
            if(rr2<=CTE.fourh2 && rr2>=ALMOSTZERO && CODE_IsFluidNotInout(code[p2])){//-Only with fluid particles but not inout particles.
              //-Wendland or Cubic Spline kernel.
			  float frx,fry,frz,wab;
			  if(tker==KERNEL_Wendland)cusph::KerGetKernelWendland(rr2,drx,dry,drz,frx,fry,frz,wab);
			  else if(tker==KERNEL_Cubic)cusph::KerGetKernelCubic(rr2,drx,dry,drz,frx,fry,frz,wab);

              const float4 velrhopp2=velrhop[p2];
              //===== Get mass and volume of particle p2 =====
              float massp2=CTE.massf;
              float volp2=massp2/velrhopp2.w;

              //===== Density and its gradient =====
              rhopp1+=massp2*wab;
              gradrhopp1.x+=massp2*frx;
              gradrhopp1.y+=massp2*fry;
              gradrhopp1.z+=massp2*frz;

              //===== Kernel values multiplied by volume =====
              const float vwab=wab*volp2;
              const float vfrx=frx*volp2;
              const float vfry=fry*volp2;
              const float vfrz=frz*volp2;

              //===== Velocity and its gradient =====
              if(computevel){
                velp1.x+=vwab*velrhopp2.x;
                velp1.y+=vwab*velrhopp2.y;
                velp1.z+=vwab*velrhopp2.z;
                gradvelp1.a11+=vfrx*velrhopp2.x;	// du/dx
                gradvelp1.a12+=vfry*velrhopp2.x;	// du/dy
                gradvelp1.a13+=vfrz*velrhopp2.x;	// du/dz
                gradvelp1.a21+=vfrx*velrhopp2.y;	// dv/dx
                gradvelp1.a22+=vfry*velrhopp2.y;	// dv/dx
                gradvelp1.a23+=vfrz*velrhopp2.y;	// dv/dx
                gradvelp1.a31+=vfrx*velrhopp2.z;	// dw/dx
                gradvelp1.a32+=vfry*velrhopp2.z;	// dw/dx
                gradvelp1.a33+=vfrz*velrhopp2.z;	// dw/dx
              }

              //===== Matrix A for correction =====
              if(sim2d){
                a_corr2.a11+=vwab; 	a_corr2.a12+=drx*vwab;	a_corr2.a13+=drz*vwab;
                a_corr2.a21+=vfrx; 	a_corr2.a22+=drx*vfrx; 	a_corr2.a23+=drz*vfrx;
                a_corr2.a31+=vfrz; 	a_corr2.a32+=drx*vfrz;	a_corr2.a33+=drz*vfrz;
              }
              else{
                a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
                a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
                a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
                a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
              }
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      float4 velrhopfinal=velrhop[p1];
      const float3 dpos=make_float3(float(pos_p1.x-posp1.x),float(pos_p1.y-posp1.y),float(pos_p1.z-posp1.z)); //-Inlet/outlet particle position - ghost node position.
      if(sim2d){
        const double determ=cumath::Determinant3x3(a_corr2);
        if(determ>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
          const tmatrix3d invacorr2=cumath::InverseMatrix3x3(a_corr2,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const float rhoghost=float(invacorr2.a11*rhopp1 + invacorr2.a12*gradrhopp1.x + invacorr2.a13*gradrhopp1.z);
            const float grx=    -float(invacorr2.a21*rhopp1 + invacorr2.a22*gradrhopp1.x + invacorr2.a23*gradrhopp1.z);
            const float grz=    -float(invacorr2.a31*rhopp1 + invacorr2.a32*gradrhopp1.x + invacorr2.a33*gradrhopp1.z);
            velrhopfinal.w=(rhoghost + grx*dpos.x + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const float velghost_x=float(invacorr2.a11*velp1.x + invacorr2.a12*gradvelp1.a11 + invacorr2.a13*gradvelp1.a13);
            const float velghost_z=float(invacorr2.a11*velp1.z + invacorr2.a12*gradvelp1.a31 + invacorr2.a13*gradvelp1.a33);
            const float a11=-float(invacorr2.a21*velp1.x + invacorr2.a22*gradvelp1.a11 + invacorr2.a23*gradvelp1.a13);
            const float a13=-float(invacorr2.a21*velp1.z + invacorr2.a22*gradvelp1.a31 + invacorr2.a23*gradvelp1.a33);
            const float a31=-float(invacorr2.a31*velp1.x + invacorr2.a32*gradvelp1.a11 + invacorr2.a33*gradvelp1.a13);
            const float a33=-float(invacorr2.a31*velp1.z + invacorr2.a32*gradvelp1.a31 + invacorr2.a33*gradvelp1.a33);
    	    velrhopfinal.x=(velghost_x + a11*dpos.x + a31*dpos.z);
    	    velrhopfinal.z=(velghost_z + a13*dpos.x + a33*dpos.z);
            velrhopfinal.y=0;
   	      }
        }
        else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr2.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr2.a11);
            velrhopfinal.z=float(velp1.z/a_corr2.a11);
            velrhopfinal.y=0;
   	      }
        }
      }
      else{
        const double determ=cumath::Determinant4x4(a_corr3);
        if(determ>=determlimit){
          const tmatrix4d invacorr3=cumath::InverseMatrix4x4(a_corr3,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const float rhoghost=float(invacorr3.a11*rhopp1 + invacorr3.a12*gradrhopp1.x + invacorr3.a13*gradrhopp1.y + invacorr3.a14*gradrhopp1.z);
            const float grx=    -float(invacorr3.a21*rhopp1 + invacorr3.a22*gradrhopp1.x + invacorr3.a23*gradrhopp1.y + invacorr3.a24*gradrhopp1.z);
            const float gry=    -float(invacorr3.a31*rhopp1 + invacorr3.a32*gradrhopp1.x + invacorr3.a33*gradrhopp1.y + invacorr3.a34*gradrhopp1.z);
            const float grz=    -float(invacorr3.a41*rhopp1 + invacorr3.a42*gradrhopp1.x + invacorr3.a43*gradrhopp1.y + invacorr3.a44*gradrhopp1.z);
            velrhopfinal.w=(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const float velghost_x=float(invacorr3.a11*velp1.x + invacorr3.a12*gradvelp1.a11 + invacorr3.a13*gradvelp1.a12 + invacorr3.a14*gradvelp1.a13);
      	    const float velghost_y=float(invacorr3.a11*velp1.y + invacorr3.a12*gradvelp1.a11 + invacorr3.a13*gradvelp1.a12 + invacorr3.a14*gradvelp1.a13);
      	    const float velghost_z=float(invacorr3.a11*velp1.z + invacorr3.a12*gradvelp1.a31 + invacorr3.a13*gradvelp1.a32 + invacorr3.a14*gradvelp1.a33);
            const float a11=      -float(invacorr3.a21*velp1.x + invacorr3.a22*gradvelp1.a11 + invacorr3.a23*gradvelp1.a12 + invacorr3.a24*gradvelp1.a13);
        	const float a12=      -float(invacorr3.a21*velp1.y + invacorr3.a22*gradvelp1.a21 + invacorr3.a23*gradvelp1.a22 + invacorr3.a24*gradvelp1.a23);
        	const float a13=      -float(invacorr3.a21*velp1.z + invacorr3.a22*gradvelp1.a31 + invacorr3.a23*gradvelp1.a32 + invacorr3.a24*gradvelp1.a33);
        	const float a21=      -float(invacorr3.a31*velp1.x + invacorr3.a32*gradvelp1.a11 + invacorr3.a33*gradvelp1.a12 + invacorr3.a34*gradvelp1.a13);
        	const float a22=      -float(invacorr3.a31*velp1.y + invacorr3.a32*gradvelp1.a21 + invacorr3.a33*gradvelp1.a22 + invacorr3.a34*gradvelp1.a23);
        	const float a23=      -float(invacorr3.a31*velp1.z + invacorr3.a32*gradvelp1.a31 + invacorr3.a33*gradvelp1.a32 + invacorr3.a34*gradvelp1.a33);
        	const float a31=      -float(invacorr3.a41*velp1.x + invacorr3.a42*gradvelp1.a11 + invacorr3.a43*gradvelp1.a12 + invacorr3.a44*gradvelp1.a13);
        	const float a32=      -float(invacorr3.a41*velp1.y + invacorr3.a42*gradvelp1.a21 + invacorr3.a43*gradvelp1.a22 + invacorr3.a44*gradvelp1.a23);
        	const float a33=      -float(invacorr3.a41*velp1.z + invacorr3.a42*gradvelp1.a31 + invacorr3.a43*gradvelp1.a32 + invacorr3.a44*gradvelp1.a33);
            velrhopfinal.x=(velghost_x + a11*dpos.x + a21*dpos.y + a31*dpos.z);
            velrhopfinal.y=(velghost_y + a12*dpos.x + a22*dpos.y + a32*dpos.z);
      	    velrhopfinal.z=(velghost_z + a13*dpos.x + a23*dpos.y + a33*dpos.z);
          }
        }
        else if(a_corr3.a11>0){ // Determinant is small but a11 is nonzero, 0th order ANGELO
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr3.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr3.a11);
            velrhopfinal.y=float(velp1.y/a_corr3.a11);
            velrhopfinal.z=float(velp1.z/a_corr3.a11);
     	  }
        }
      }
      velrhop[p1]=velrhopfinal;
    }
  }
}


//------------------------------------------------------------------------------
/// Perform interaction between ghost inlet/outlet nodes and fluid particles. GhostNodes-Fluid
/// Realiza interaccion entre ghost inlet/outlet nodes y particulas de fluido. GhostNodes-Fluid
//------------------------------------------------------------------------------
template<bool sim2d,TpKernel tker> __global__ void KerInteractionInOutExtrap_FastSingle
  (unsigned inoutcount,const int *inoutpart,const byte *cfgzone,byte computerhopmask,byte computevelmask
  ,const float4 *planes,const float* width,const float3 *dirdata,float determlimit
  ,int hdiv,int4 nc,unsigned cellfluid,const int2 *begincell,int3 cellzero
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<inoutcount){
    const unsigned p1=inoutpart[cp];
    const byte izone=byte(CODE_GetIzoneFluidInout(code[p1]));
    const byte cfg=cfgzone[izone];
    const bool computerhop=((cfg&computerhopmask)!=0);
    const bool computevel= ((cfg&computevelmask )!=0);
    if(computerhop || computevel){
      //-Calculates ghost node position.
      double3 pos_p1=make_double3(posxy[p1].x,posxy[p1].y,posz[p1]);
      if(CODE_IsPeriodic(code[p1]))pos_p1=KerInteraction_PosNoPeriodic(pos_p1);
      const double displane=cumath::DistPlane(planes[izone],pos_p1)*2;
      const float3 rdirdata=dirdata[izone];
      const double3 posp1=make_double3(pos_p1.x+displane*rdirdata.x, pos_p1.y+displane*rdirdata.y, pos_p1.z+displane*rdirdata.z); //-Ghost node position.

      //-Initializes variables for calculation.
      float rhopp1=0;
      float3 gradrhopp1=make_float3(0,0,0);
      float3 velp1=make_float3(0,0,0);
      tmatrix3f gradvelp1; cumath::Tmatrix3fReset(gradvelp1); //-Only for velocity.
      tmatrix3f a_corr2; if(sim2d) cumath::Tmatrix3fReset(a_corr2); //-Only for 2D.
      tmatrix4f a_corr3; if(!sim2d)cumath::Tmatrix4fReset(a_corr3); //-Only for 3D.

      //-Obtains interaction limits.
      int cxini,cxfin,yini,yfin,zini,zfin;
      cusph::KerGetInteractionCells(posp1.x,posp1.y,posp1.z,hdiv,nc,cellzero,cxini,cxfin,yini,yfin,zini,zfin);

      //-Interaction with fluids.
      for(int z=zini;z<zfin;z++){
        int zmod=(nc.w)*z+cellfluid; //-The sum showing where fluid cells start. | Le suma donde empiezan las celdas de fluido.
        for(int y=yini;y<yfin;y++){
          int ymod=zmod+nc.x*y;
          unsigned pini,pfin=0;
          for(int x=cxini;x<cxfin;x++){
            int2 cbeg=begincell[x+ymod];
            if(cbeg.y){
              if(!pfin)pini=cbeg.x;
              pfin=cbeg.y;
            }
          }
          if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
            const double2 p2xy=posxy[p2];
            const float drx=float(posp1.x-p2xy.x);
            const float dry=float(posp1.y-p2xy.y);
            const float drz=float(posp1.z-posz[p2]);
            const float rr2=drx*drx+dry*dry+drz*drz;
            if(rr2<=CTE.fourh2 && rr2>=ALMOSTZERO && CODE_IsFluidNotInout(code[p2])){//-Only with fluid particles but not inout particles.
              //-Wendland or Cubic Spline kernel.
			  float frx,fry,frz,wab;
			  if(tker==KERNEL_Wendland)cusph::KerGetKernelWendland(rr2,drx,dry,drz,frx,fry,frz,wab);
			  else if(tker==KERNEL_Cubic)cusph::KerGetKernelCubic(rr2,drx,dry,drz,frx,fry,frz,wab);

              const float4 velrhopp2=velrhop[p2];
              //===== Get mass and volume of particle p2 =====
              float massp2=CTE.massf;
              float volp2=massp2/velrhopp2.w;

              //===== Density and its gradient =====
              rhopp1+=massp2*wab;
              gradrhopp1.x+=massp2*frx;
              gradrhopp1.y+=massp2*fry;
              gradrhopp1.z+=massp2*frz;

              //===== Kernel values multiplied by volume =====
              const float vwab=wab*volp2;
              const float vfrx=frx*volp2;
              const float vfry=fry*volp2;
              const float vfrz=frz*volp2;

              //===== Velocity and its gradient =====
              if(computevel){
                velp1.x+=vwab*velrhopp2.x;
                velp1.y+=vwab*velrhopp2.y;
                velp1.z+=vwab*velrhopp2.z;
                gradvelp1.a11+=vfrx*velrhopp2.x;	// du/dx
                gradvelp1.a12+=vfry*velrhopp2.x;	// du/dy
                gradvelp1.a13+=vfrz*velrhopp2.x;	// du/dz
                gradvelp1.a21+=vfrx*velrhopp2.y;	// dv/dx
                gradvelp1.a22+=vfry*velrhopp2.y;	// dv/dx
                gradvelp1.a23+=vfrz*velrhopp2.y;	// dv/dx
                gradvelp1.a31+=vfrx*velrhopp2.z;	// dw/dx
                gradvelp1.a32+=vfry*velrhopp2.z;	// dw/dx
                gradvelp1.a33+=vfrz*velrhopp2.z;	// dw/dx
              }

              //===== Matrix A for correction =====
              if(sim2d){
                a_corr2.a11+=vwab; 	a_corr2.a12+=drx*vwab;	a_corr2.a13+=drz*vwab;
                a_corr2.a21+=vfrx; 	a_corr2.a22+=drx*vfrx; 	a_corr2.a23+=drz*vfrx;
                a_corr2.a31+=vfrz; 	a_corr2.a32+=drx*vfrz;	a_corr2.a33+=drz*vfrz;
              }
              else{
                a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
                a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
                a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
                a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
              }
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      float4 velrhopfinal=velrhop[p1];
      const float3 dpos=make_float3(float(pos_p1.x-posp1.x),float(pos_p1.y-posp1.y),float(pos_p1.z-posp1.z)); //-Inlet/outlet particle position - ghost node position.
      if(sim2d){
        const double determ=cumath::Determinant3x3dbl(a_corr2);
        if(determ>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
          const tmatrix3f invacorr2=cumath::InverseMatrix3x3dbl(a_corr2,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const float rhoghost=float(invacorr2.a11*rhopp1 + invacorr2.a12*gradrhopp1.x + invacorr2.a13*gradrhopp1.z);
            const float grx=    -float(invacorr2.a21*rhopp1 + invacorr2.a22*gradrhopp1.x + invacorr2.a23*gradrhopp1.z);
            const float grz=    -float(invacorr2.a31*rhopp1 + invacorr2.a32*gradrhopp1.x + invacorr2.a33*gradrhopp1.z);
            velrhopfinal.w=(rhoghost + grx*dpos.x + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const float velghost_x=float(invacorr2.a11*velp1.x + invacorr2.a12*gradvelp1.a11 + invacorr2.a13*gradvelp1.a13);
            const float velghost_z=float(invacorr2.a11*velp1.z + invacorr2.a12*gradvelp1.a31 + invacorr2.a13*gradvelp1.a33);
            const float a11=-float(invacorr2.a21*velp1.x + invacorr2.a22*gradvelp1.a11 + invacorr2.a23*gradvelp1.a13);
            const float a13=-float(invacorr2.a21*velp1.z + invacorr2.a22*gradvelp1.a31 + invacorr2.a23*gradvelp1.a33);
            const float a31=-float(invacorr2.a31*velp1.x + invacorr2.a32*gradvelp1.a11 + invacorr2.a33*gradvelp1.a13);
            const float a33=-float(invacorr2.a31*velp1.z + invacorr2.a32*gradvelp1.a31 + invacorr2.a33*gradvelp1.a33);
    	    velrhopfinal.x=(velghost_x + a11*dpos.x + a31*dpos.z);
    	    velrhopfinal.z=(velghost_z + a13*dpos.x + a33*dpos.z);
            velrhopfinal.y=0;
   	      }
        }
        else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr2.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr2.a11);
            velrhopfinal.z=float(velp1.z/a_corr2.a11);
            velrhopfinal.y=0;
   	      }
        }
      }
      else{
        const double determ=cumath::Determinant4x4dbl(a_corr3);
        if(determ>=determlimit){
          const tmatrix4f invacorr3=cumath::InverseMatrix4x4dbl(a_corr3,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const float rhoghost=float(invacorr3.a11*rhopp1 + invacorr3.a12*gradrhopp1.x + invacorr3.a13*gradrhopp1.y + invacorr3.a14*gradrhopp1.z);
            const float grx=    -float(invacorr3.a21*rhopp1 + invacorr3.a22*gradrhopp1.x + invacorr3.a23*gradrhopp1.y + invacorr3.a24*gradrhopp1.z);
            const float gry=    -float(invacorr3.a31*rhopp1 + invacorr3.a32*gradrhopp1.x + invacorr3.a33*gradrhopp1.y + invacorr3.a34*gradrhopp1.z);
            const float grz=    -float(invacorr3.a41*rhopp1 + invacorr3.a42*gradrhopp1.x + invacorr3.a43*gradrhopp1.y + invacorr3.a44*gradrhopp1.z);
            velrhopfinal.w=(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const float velghost_x=float(invacorr3.a11*velp1.x + invacorr3.a12*gradvelp1.a11 + invacorr3.a13*gradvelp1.a12 + invacorr3.a14*gradvelp1.a13);
      	    const float velghost_y=float(invacorr3.a11*velp1.y + invacorr3.a12*gradvelp1.a11 + invacorr3.a13*gradvelp1.a12 + invacorr3.a14*gradvelp1.a13);
      	    const float velghost_z=float(invacorr3.a11*velp1.z + invacorr3.a12*gradvelp1.a31 + invacorr3.a13*gradvelp1.a32 + invacorr3.a14*gradvelp1.a33);
            const float a11=      -float(invacorr3.a21*velp1.x + invacorr3.a22*gradvelp1.a11 + invacorr3.a23*gradvelp1.a12 + invacorr3.a24*gradvelp1.a13);
        	const float a12=      -float(invacorr3.a21*velp1.y + invacorr3.a22*gradvelp1.a21 + invacorr3.a23*gradvelp1.a22 + invacorr3.a24*gradvelp1.a23);
        	const float a13=      -float(invacorr3.a21*velp1.z + invacorr3.a22*gradvelp1.a31 + invacorr3.a23*gradvelp1.a32 + invacorr3.a24*gradvelp1.a33);
        	const float a21=      -float(invacorr3.a31*velp1.x + invacorr3.a32*gradvelp1.a11 + invacorr3.a33*gradvelp1.a12 + invacorr3.a34*gradvelp1.a13);
        	const float a22=      -float(invacorr3.a31*velp1.y + invacorr3.a32*gradvelp1.a21 + invacorr3.a33*gradvelp1.a22 + invacorr3.a34*gradvelp1.a23);
        	const float a23=      -float(invacorr3.a31*velp1.z + invacorr3.a32*gradvelp1.a31 + invacorr3.a33*gradvelp1.a32 + invacorr3.a34*gradvelp1.a33);
        	const float a31=      -float(invacorr3.a41*velp1.x + invacorr3.a42*gradvelp1.a11 + invacorr3.a43*gradvelp1.a12 + invacorr3.a44*gradvelp1.a13);
        	const float a32=      -float(invacorr3.a41*velp1.y + invacorr3.a42*gradvelp1.a21 + invacorr3.a43*gradvelp1.a22 + invacorr3.a44*gradvelp1.a23);
        	const float a33=      -float(invacorr3.a41*velp1.z + invacorr3.a42*gradvelp1.a31 + invacorr3.a43*gradvelp1.a32 + invacorr3.a44*gradvelp1.a33);
            velrhopfinal.x=(velghost_x + a11*dpos.x + a21*dpos.y + a31*dpos.z);
            velrhopfinal.y=(velghost_y + a12*dpos.x + a22*dpos.y + a32*dpos.z);
      	    velrhopfinal.z=(velghost_z + a13*dpos.x + a23*dpos.y + a33*dpos.z);
          }
        }
        else if(a_corr3.a11>0){ // Determinant is small but a11 is nonzero, 0th order ANGELO
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr3.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr3.a11);
            velrhopfinal.y=float(velp1.y/a_corr3.a11);
            velrhopfinal.z=float(velp1.z/a_corr3.a11);
     	  }
        }
      }
      velrhop[p1]=velrhopfinal;
    }
  }
}

//==============================================================================
/// Perform interaction between ghost inlet/outlet nodes and fluid particles. GhostNodes-Fluid
/// Realiza interaccion entre ghost inlet/outlet nodes y particulas de fluido. GhostNodes-Fluid
//==============================================================================
void Interaction_InOutExtrap(byte doublemode,bool simulate2d,TpKernel tkernel,TpCellMode cellmode
  ,unsigned inoutcount,const int *inoutpart,const byte *cfgzone,byte computerhopmask,byte computevelmask
  ,const float4 *planes,const float* width,const float3 *dirdata,float determlimit
  ,tuint3 ncells,const int2 *begincell,tuint3 cellmin
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  //-Executes particle interactions.
  const int hdiv=(cellmode==CELLMODE_H? 2: 1);
  const int4 nc=make_int4(int(ncells.x),int(ncells.y),int(ncells.z),int(ncells.x*ncells.y));
  const unsigned cellfluid=nc.w*nc.z+1;
  const int3 cellzero=make_int3(cellmin.x,cellmin.y,cellmin.z);
  //-Interaction GhostBoundaryNodes-Fluid.
  if(inoutcount){
    const unsigned bsize=128;
    dim3 sgrid=cusph::GetGridSize(inoutcount,bsize);
    if(doublemode==1){
      if(simulate2d){ const bool sim2d=true;
        if(tkernel==KERNEL_Wendland)KerInteractionInOutExtrap_FastSingle<sim2d,KERNEL_Wendland> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionInOutExtrap_FastSingle<sim2d,KERNEL_Cubic>    <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }else{          const bool sim2d=false;
        if(tkernel==KERNEL_Wendland)KerInteractionInOutExtrap_FastSingle<sim2d,KERNEL_Wendland> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionInOutExtrap_FastSingle<sim2d,KERNEL_Cubic>    <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }
    }
    else if(doublemode==2){
      if(simulate2d){ const bool sim2d=true;
        if(tkernel==KERNEL_Wendland)KerInteractionInOutExtrap_Single<sim2d,KERNEL_Wendland> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionInOutExtrap_Single<sim2d,KERNEL_Cubic>    <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }else{          const bool sim2d=false;
        if(tkernel==KERNEL_Wendland)KerInteractionInOutExtrap_Single<sim2d,KERNEL_Wendland> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionInOutExtrap_Single<sim2d,KERNEL_Cubic>    <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }
    }
    else if(doublemode==3){
      if(simulate2d){ const bool sim2d=true;
        if(tkernel==KERNEL_Wendland)KerInteractionInOutExtrap_Double<sim2d,KERNEL_Wendland> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionInOutExtrap_Double<sim2d,KERNEL_Cubic>    <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }else{          const bool sim2d=false;
        if(tkernel==KERNEL_Wendland)KerInteractionInOutExtrap_Double<sim2d,KERNEL_Wendland> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionInOutExtrap_Double<sim2d,KERNEL_Cubic>    <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }
    }
  }
}


//##############################################################################
//# Kernels to extrapolate rhop on boundary particles (JSphBoundCorr).
//# Kernels para extrapolar rhop en las particulas de contorno (JSphBoundCorr).
//##############################################################################
//------------------------------------------------------------------------------
/// Perform interaction between ghost node of selected boundary and fluid.
//------------------------------------------------------------------------------
template<bool sim2d,TpKernel tker> __global__ void KerInteractionBoundCorr_Double
  (unsigned npb,typecode boundcode,float4 plane,float3 direction,float determlimit
  ,int hdiv,int4 nc,unsigned cellfluid,const int2 *begincell,int3 cellzero
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  const unsigned p1=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p1<npb && CODE_GetTypeAndValue(code[p1])==boundcode){
    float rhopfinal=FLT_MAX;
    //-Calculates ghost node position.
    double3 pos_p1=make_double3(posxy[p1].x,posxy[p1].y,posz[p1]);
    if(CODE_IsPeriodic(code[p1]))pos_p1=KerInteraction_PosNoPeriodic(pos_p1);
    const double displane=cumath::DistPlane(plane,pos_p1)*2;
    if(displane<=CTE.h*4.f){
      const double3 posp1=make_double3(pos_p1.x+displane*direction.x, pos_p1.y+displane*direction.y, pos_p1.z+displane*direction.z); //-Ghost node position.
      //-Initializes variables for calculation.
      double rhopp1=0;
      double3 gradrhopp1=make_double3(0,0,0);
      tmatrix3d a_corr2; if(sim2d) cumath::Tmatrix3dReset(a_corr2); //-Only for 2D.
      tmatrix4d a_corr3; if(!sim2d)cumath::Tmatrix4dReset(a_corr3); //-Only for 3D.

      //-Obtains interaction limits.
      int cxini,cxfin,yini,yfin,zini,zfin;
      cusph::KerGetInteractionCells(posp1.x,posp1.y,posp1.z,hdiv,nc,cellzero,cxini,cxfin,yini,yfin,zini,zfin);

      //-Interaction with fluids.
      for(int z=zini;z<zfin;z++){
        int zmod=(nc.w)*z+cellfluid; //-The sum showing where fluid cells start. | Le suma donde empiezan las celdas de fluido.
        for(int y=yini;y<yfin;y++){
          int ymod=zmod+nc.x*y;
          unsigned pini,pfin=0;
          for(int x=cxini;x<cxfin;x++){
            int2 cbeg=begincell[x+ymod];
            if(cbeg.y){
              if(!pfin)pini=cbeg.x;
              pfin=cbeg.y;
            }
          }
          if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
            const double2 p2xy=posxy[p2];
            const double drx=double(posp1.x-p2xy.x);
            const double dry=double(posp1.y-p2xy.y);
            const double drz=double(posp1.z-posz[p2]);
            const double rr2=drx*drx+dry*dry+drz*drz;
            if(rr2<=CTE.fourh2 && rr2>=ALMOSTZERO && CODE_IsFluid(code[p2])){//-Only with fluid particles (including inout).
              //-Wendland or Cubic Spline kernel.
              float ffrx,ffry,ffrz,fwab;
              if(tker==KERNEL_Wendland)cusph::KerGetKernelWendland(float(rr2),float(drx),float(dry),float(drz),ffrx,ffry,ffrz,fwab);
              else if(tker==KERNEL_Cubic)cusph::KerGetKernelCubic(float(rr2),float(drx),float(dry),float(drz),ffrx,ffry,ffrz,fwab);
              const double frx=ffrx,fry=ffry,frz=ffrz,wab=fwab;

              //===== Get mass and volume of particle p2 =====
              const double massp2=CTE.massf;
              const double volp2=massp2/double(velrhop[p2].w);

              //===== Density and its gradient =====
              rhopp1+=massp2*wab;
              gradrhopp1.x+=massp2*frx;
              gradrhopp1.y+=massp2*fry;
              gradrhopp1.z+=massp2*frz;

              //===== Kernel values multiplied by volume =====
              const double vwab=wab*volp2;
              const double vfrx=frx*volp2;
              const double vfry=fry*volp2;
              const double vfrz=frz*volp2;

              //===== Matrix A for correction =====
              if(sim2d){
                a_corr2.a11+=vwab;  a_corr2.a12+=drx*vwab;  a_corr2.a13+=drz*vwab;
                a_corr2.a21+=vfrx;  a_corr2.a22+=drx*vfrx;  a_corr2.a23+=drz*vfrx;
                a_corr2.a31+=vfrz;  a_corr2.a32+=drx*vfrz;  a_corr2.a33+=drz*vfrz;
              }
              else{
                a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
                a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
                a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
                a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
              }
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      const double3 dpos=make_double3(pos_p1.x-posp1.x, pos_p1.y-posp1.y, pos_p1.z-posp1.z); //-Boundary particle position - ghost node position.
      if(sim2d){
        const double determ=cumath::Determinant3x3(a_corr2);
        if(determ>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
          const tmatrix3d invacorr2=cumath::InverseMatrix3x3(a_corr2,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          const double rhoghost=rhopp1*invacorr2.a11 + gradrhopp1.x*invacorr2.a12 + gradrhopp1.z*invacorr2.a13;
          const double grx=-(rhopp1*invacorr2.a21 + gradrhopp1.x*invacorr2.a22 + gradrhopp1.z*invacorr2.a23);
          const double grz=-(rhopp1*invacorr2.a31 + gradrhopp1.x*invacorr2.a32 + gradrhopp1.z*invacorr2.a33);
          rhopfinal=float(rhoghost + grx*dpos.x + grz*dpos.z);
        }
        else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          rhopfinal=float(rhopp1/a_corr2.a11);
        }
      }
      else{
        const double determ=cumath::Determinant4x4(a_corr3);
        if(determ>=determlimit){
          const tmatrix4d invacorr3=cumath::InverseMatrix4x4(a_corr3,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          const double rhoghost=rhopp1*invacorr3.a11 + gradrhopp1.x*invacorr3.a12 + gradrhopp1.y*invacorr3.a13 + gradrhopp1.z*invacorr3.a14;
          const double grx=   -(rhopp1*invacorr3.a21 + gradrhopp1.x*invacorr3.a22 + gradrhopp1.y*invacorr3.a23 + gradrhopp1.z*invacorr3.a24);
          const double gry=   -(rhopp1*invacorr3.a31 + gradrhopp1.x*invacorr3.a32 + gradrhopp1.y*invacorr3.a33 + gradrhopp1.z*invacorr3.a34);
          const double grz=   -(rhopp1*invacorr3.a41 + gradrhopp1.x*invacorr3.a42 + gradrhopp1.y*invacorr3.a43 + gradrhopp1.z*invacorr3.a44);
          rhopfinal=float(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
        }
        else if(a_corr3.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          rhopfinal=float(rhopp1/a_corr3.a11);
        }
      }
    }
    velrhop[p1].w=(rhopfinal!=FLT_MAX? rhopfinal: CTE.rhopzero);
  }
}

//------------------------------------------------------------------------------
/// Perform interaction between ghost node of selected boundary and fluid.
//------------------------------------------------------------------------------
template<bool sim2d,TpKernel tker> __global__ void KerInteractionBoundCorr_Single
  (unsigned npb,typecode boundcode,float4 plane,float3 direction,float determlimit
  ,int hdiv,int4 nc,unsigned cellfluid,const int2 *begincell,int3 cellzero
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  const unsigned p1=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p1<npb && CODE_GetTypeAndValue(code[p1])==boundcode){
    float rhopfinal=FLT_MAX;
    //-Calculates ghost node position.
    double3 pos_p1=make_double3(posxy[p1].x,posxy[p1].y,posz[p1]);
    if(CODE_IsPeriodic(code[p1]))pos_p1=KerInteraction_PosNoPeriodic(pos_p1);
    const double displane=cumath::DistPlane(plane,pos_p1)*2;
    if(displane<=CTE.h*4.f){
      const double3 posp1=make_double3(pos_p1.x+displane*direction.x, pos_p1.y+displane*direction.y, pos_p1.z+displane*direction.z); //-Ghost node position.
      //-Initializes variables for calculation.
      float rhopp1=0;
      float3 gradrhopp1=make_float3(0,0,0);
      tmatrix3d a_corr2; if(sim2d) cumath::Tmatrix3dReset(a_corr2); //-Only for 2D.
      tmatrix4d a_corr3; if(!sim2d)cumath::Tmatrix4dReset(a_corr3); //-Only for 3D.

      //-Obtains interaction limits.
      int cxini,cxfin,yini,yfin,zini,zfin;
      cusph::KerGetInteractionCells(posp1.x,posp1.y,posp1.z,hdiv,nc,cellzero,cxini,cxfin,yini,yfin,zini,zfin);

      //-Interaction with fluids.
      for(int z=zini;z<zfin;z++){
        int zmod=(nc.w)*z+cellfluid; //-The sum showing where fluid cells start. | Le suma donde empiezan las celdas de fluido.
        for(int y=yini;y<yfin;y++){
          int ymod=zmod+nc.x*y;
          unsigned pini,pfin=0;
          for(int x=cxini;x<cxfin;x++){
            int2 cbeg=begincell[x+ymod];
            if(cbeg.y){
              if(!pfin)pini=cbeg.x;
              pfin=cbeg.y;
            }
          }
          if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
            const double2 p2xy=posxy[p2];
            const float drx=float(posp1.x-p2xy.x);
            const float dry=float(posp1.y-p2xy.y);
            const float drz=float(posp1.z-posz[p2]);
            const float rr2=drx*drx+dry*dry+drz*drz;
            if(rr2<=CTE.fourh2 && rr2>=ALMOSTZERO && CODE_IsFluid(code[p2])){//-Only with fluid particles (including inout).
              //-Wendland or Cubic Spline kernel.
              float frx,fry,frz,wab;
              if(tker==KERNEL_Wendland)cusph::KerGetKernelWendland(rr2,drx,dry,drz,frx,fry,frz,wab);
              else if(tker==KERNEL_Cubic)cusph::KerGetKernelCubic(rr2,drx,dry,drz,frx,fry,frz,wab);

              //===== Get mass and volume of particle p2 =====
              float massp2=CTE.massf;
              const float volp2=massp2/velrhop[p2].w;

              //===== Density and its gradient =====
              rhopp1+=massp2*wab;
              gradrhopp1.x+=massp2*frx;
              gradrhopp1.y+=massp2*fry;
              gradrhopp1.z+=massp2*frz;

              //===== Kernel values multiplied by volume =====
              const float vwab=wab*volp2;
              const float vfrx=frx*volp2;
              const float vfry=fry*volp2;
              const float vfrz=frz*volp2;

              //===== Matrix A for correction =====
              if(sim2d){
                a_corr2.a11+=vwab;  a_corr2.a12+=drx*vwab;  a_corr2.a13+=drz*vwab;
                a_corr2.a21+=vfrx;  a_corr2.a22+=drx*vfrx;  a_corr2.a23+=drz*vfrx;
                a_corr2.a31+=vfrz;  a_corr2.a32+=drx*vfrz;  a_corr2.a33+=drz*vfrz;
              }
              else{
                a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
                a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
                a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
                a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
              }
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      const float3 dpos=make_float3(float(pos_p1.x-posp1.x),float(pos_p1.y-posp1.y),float(pos_p1.z-posp1.z)); //-Boundary particle position - ghost node position.
      if(sim2d){
        const double determ=cumath::Determinant3x3(a_corr2);
        if(determ>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
          const tmatrix3d invacorr2=cumath::InverseMatrix3x3(a_corr2,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          const float rhoghost=float(invacorr2.a11*rhopp1 + invacorr2.a12*gradrhopp1.x + invacorr2.a13*gradrhopp1.z);
          const float grx=    -float(invacorr2.a21*rhopp1 + invacorr2.a22*gradrhopp1.x + invacorr2.a23*gradrhopp1.z);
          const float grz=    -float(invacorr2.a31*rhopp1 + invacorr2.a32*gradrhopp1.x + invacorr2.a33*gradrhopp1.z);
          rhopfinal=(rhoghost + grx*dpos.x + grz*dpos.z);
        }
        else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          rhopfinal=float(rhopp1/a_corr2.a11);
        }
      }
      else{
        const double determ=cumath::Determinant4x4(a_corr3);
        if(determ>=determlimit){
          const tmatrix4d invacorr3=cumath::InverseMatrix4x4(a_corr3,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          const float rhoghost=float(invacorr3.a11*rhopp1 + invacorr3.a12*gradrhopp1.x + invacorr3.a13*gradrhopp1.y + invacorr3.a14*gradrhopp1.z);
          const float grx=    -float(invacorr3.a21*rhopp1 + invacorr3.a22*gradrhopp1.x + invacorr3.a23*gradrhopp1.y + invacorr3.a24*gradrhopp1.z);
          const float gry=    -float(invacorr3.a31*rhopp1 + invacorr3.a32*gradrhopp1.x + invacorr3.a33*gradrhopp1.y + invacorr3.a34*gradrhopp1.z);
          const float grz=    -float(invacorr3.a41*rhopp1 + invacorr3.a42*gradrhopp1.x + invacorr3.a43*gradrhopp1.y + invacorr3.a44*gradrhopp1.z);
          rhopfinal=(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
        }
        else if(a_corr3.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          rhopfinal=float(rhopp1/a_corr3.a11);
        }
      }
    }
    velrhop[p1].w=(rhopfinal!=FLT_MAX? rhopfinal: CTE.rhopzero);
  }
}


//------------------------------------------------------------------------------
/// Perform interaction between ghost node of selected boundary and fluid.
//------------------------------------------------------------------------------
template<bool sim2d,TpKernel tker> __global__ void KerInteractionBoundCorr_FastSingle
  (unsigned npb,typecode boundcode,float4 plane,float3 direction,float determlimit
  ,int hdiv,int4 nc,unsigned cellfluid,const int2 *begincell,int3 cellzero
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  const unsigned p1=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p1<npb && CODE_GetTypeAndValue(code[p1])==boundcode){
    float rhopfinal=FLT_MAX;
    //-Calculates ghost node position.
    double3 pos_p1=make_double3(posxy[p1].x,posxy[p1].y,posz[p1]);
    if(CODE_IsPeriodic(code[p1]))pos_p1=KerInteraction_PosNoPeriodic(pos_p1);
    const double displane=cumath::DistPlane(plane,pos_p1)*2;
    if(displane<=CTE.h*4.f){
      const double3 posp1=make_double3(pos_p1.x+displane*direction.x, pos_p1.y+displane*direction.y, pos_p1.z+displane*direction.z); //-Ghost node position.
      //-Initializes variables for calculation.
      float rhopp1=0;
      float3 gradrhopp1=make_float3(0,0,0);
      tmatrix3f a_corr2; if(sim2d) cumath::Tmatrix3fReset(a_corr2); //-Only for 2D.
      tmatrix4f a_corr3; if(!sim2d)cumath::Tmatrix4fReset(a_corr3); //-Only for 3D.

      //-Obtains interaction limits.
      int cxini,cxfin,yini,yfin,zini,zfin;
      cusph::KerGetInteractionCells(posp1.x,posp1.y,posp1.z,hdiv,nc,cellzero,cxini,cxfin,yini,yfin,zini,zfin);

      //-Interaction with fluids.
      for(int z=zini;z<zfin;z++){
        int zmod=(nc.w)*z+cellfluid; //-The sum showing where fluid cells start. | Le suma donde empiezan las celdas de fluido.
        for(int y=yini;y<yfin;y++){
          int ymod=zmod+nc.x*y;
          unsigned pini,pfin=0;
          for(int x=cxini;x<cxfin;x++){
            int2 cbeg=begincell[x+ymod];
            if(cbeg.y){
              if(!pfin)pini=cbeg.x;
              pfin=cbeg.y;
            }
          }
          if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
            const double2 p2xy=posxy[p2];
            const float drx=float(posp1.x-p2xy.x);
            const float dry=float(posp1.y-p2xy.y);
            const float drz=float(posp1.z-posz[p2]);
            const float rr2=drx*drx+dry*dry+drz*drz;
            if(rr2<=CTE.fourh2 && rr2>=ALMOSTZERO && CODE_IsFluid(code[p2])){//-Only with fluid particles (including inout).
              //-Wendland or Cubic Spline kernel.
              float frx,fry,frz,wab;
              if(tker==KERNEL_Wendland)cusph::KerGetKernelWendland(rr2,drx,dry,drz,frx,fry,frz,wab);
              else if(tker==KERNEL_Cubic)cusph::KerGetKernelCubic(rr2,drx,dry,drz,frx,fry,frz,wab);

              //===== Get mass and volume of particle p2 =====
              float massp2=CTE.massf;
              const float volp2=massp2/velrhop[p2].w;

              //===== Density and its gradient =====
              rhopp1+=massp2*wab;
              gradrhopp1.x+=massp2*frx;
              gradrhopp1.y+=massp2*fry;
              gradrhopp1.z+=massp2*frz;

              //===== Kernel values multiplied by volume =====
              const float vwab=wab*volp2;
              const float vfrx=frx*volp2;
              const float vfry=fry*volp2;
              const float vfrz=frz*volp2;

              //===== Matrix A for correction =====
              if(sim2d){
                a_corr2.a11+=vwab;  a_corr2.a12+=drx*vwab;  a_corr2.a13+=drz*vwab;
                a_corr2.a21+=vfrx;  a_corr2.a22+=drx*vfrx;  a_corr2.a23+=drz*vfrx;
                a_corr2.a31+=vfrz;  a_corr2.a32+=drx*vfrz;  a_corr2.a33+=drz*vfrz;
              }
              else{
                a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
                a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
                a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
                a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
              }
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      const float3 dpos=make_float3(float(pos_p1.x-posp1.x),float(pos_p1.y-posp1.y),float(pos_p1.z-posp1.z)); //-Boundary particle position - ghost node position.
      if(sim2d){
//if(-a_corr2.a22-a_corr2.a33>0.9){ //-Suggested by Renato...
        const double determ=cumath::Determinant3x3dbl(a_corr2);
        if(determ>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
          const tmatrix3f invacorr2=cumath::InverseMatrix3x3dbl(a_corr2,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          const float rhoghost=float(invacorr2.a11*rhopp1 + invacorr2.a12*gradrhopp1.x + invacorr2.a13*gradrhopp1.z);
          const float grx=    -float(invacorr2.a21*rhopp1 + invacorr2.a22*gradrhopp1.x + invacorr2.a23*gradrhopp1.z);
          const float grz=    -float(invacorr2.a31*rhopp1 + invacorr2.a32*gradrhopp1.x + invacorr2.a33*gradrhopp1.z);
          rhopfinal=(rhoghost + grx*dpos.x + grz*dpos.z);
        }
        else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          rhopfinal=float(rhopp1/a_corr2.a11);
        }
//}
      }
      else{
        const double determ=cumath::Determinant4x4dbl(a_corr3);
        if(determ>=determlimit){
          const tmatrix4f invacorr3=cumath::InverseMatrix4x4dbl(a_corr3,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          const float rhoghost=float(invacorr3.a11*rhopp1 + invacorr3.a12*gradrhopp1.x + invacorr3.a13*gradrhopp1.y + invacorr3.a14*gradrhopp1.z);
          const float grx=    -float(invacorr3.a21*rhopp1 + invacorr3.a22*gradrhopp1.x + invacorr3.a23*gradrhopp1.y + invacorr3.a24*gradrhopp1.z);
          const float gry=    -float(invacorr3.a31*rhopp1 + invacorr3.a32*gradrhopp1.x + invacorr3.a33*gradrhopp1.y + invacorr3.a34*gradrhopp1.z);
          const float grz=    -float(invacorr3.a41*rhopp1 + invacorr3.a42*gradrhopp1.x + invacorr3.a43*gradrhopp1.y + invacorr3.a44*gradrhopp1.z);
          rhopfinal=(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
        }
        else if(a_corr3.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          rhopfinal=float(rhopp1/a_corr3.a11);
        }
      }
    }
    velrhop[p1].w=(rhopfinal!=FLT_MAX? rhopfinal: CTE.rhopzero);
  }
}

//==============================================================================
/// Perform interaction between ghost node of selected boundary and fluid.
//==============================================================================
void Interaction_BoundCorr(byte doublemode,bool simulate2d,TpKernel tkernel,TpCellMode cellmode
  ,unsigned npbok,typecode boundcode,tfloat4 plane,tfloat3 direction,float determlimit
  ,tuint3 ncells,const int2 *begincell,tuint3 cellmin
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  //-Executes particle interactions.
  const int hdiv=(cellmode==CELLMODE_H? 2: 1);
  const int4 nc=make_int4(int(ncells.x),int(ncells.y),int(ncells.z),int(ncells.x*ncells.y));
  const unsigned cellfluid=nc.w*nc.z+1;
  const int3 cellzero=make_int3(cellmin.x,cellmin.y,cellmin.z);
  //-Interaction GhostBoundaryNodes-Fluid.
  if(npbok){
    const unsigned bsbound=128;
    dim3 sgridb=cusph::GetGridSize(npbok,bsbound);
    if(doublemode==1){
      if(simulate2d){ const bool sim2d=true;
        if(tkernel==KERNEL_Wendland)KerInteractionBoundCorr_FastSingle<sim2d,KERNEL_Wendland> <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionBoundCorr_FastSingle<sim2d,KERNEL_Cubic>    <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }else{          const bool sim2d=false;
        if(tkernel==KERNEL_Wendland)KerInteractionBoundCorr_FastSingle<sim2d,KERNEL_Wendland> <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionBoundCorr_FastSingle<sim2d,KERNEL_Cubic>    <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }
    }
    else if(doublemode==2){
      if(simulate2d){ const bool sim2d=true;
        if(tkernel==KERNEL_Wendland)KerInteractionBoundCorr_Single<sim2d,KERNEL_Wendland> <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionBoundCorr_Single<sim2d,KERNEL_Cubic>    <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }else{          const bool sim2d=false;
        if(tkernel==KERNEL_Wendland)KerInteractionBoundCorr_Single<sim2d,KERNEL_Wendland> <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionBoundCorr_Single<sim2d,KERNEL_Cubic>    <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }
    }
    else if(doublemode==3){
      if(simulate2d){ const bool sim2d=true;
        if(tkernel==KERNEL_Wendland)KerInteractionBoundCorr_Double<sim2d,KERNEL_Wendland> <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionBoundCorr_Double<sim2d,KERNEL_Cubic>    <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }else{          const bool sim2d=false;
        if(tkernel==KERNEL_Wendland)KerInteractionBoundCorr_Double<sim2d,KERNEL_Wendland> <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
        if(tkernel==KERNEL_Cubic)   KerInteractionBoundCorr_Double<sim2d,KERNEL_Cubic>    <<<sgridb,bsbound>>> (npbok,boundcode,Float4(plane),Float3(direction),determlimit,hdiv,nc,cellfluid,begincell,cellzero,posxy,posz,code,idp,velrhop);
      }
    }
  }
}


//##############################################################################
//# Kernels to interpolate velocity (JSphInOutGridDataTime).
//# Kernels para interpolar valores de velocidad (JSphInOutGridDataTime).
//##############################################################################
//------------------------------------------------------------------------------
/// Interpolate data between time0 and time1.
//------------------------------------------------------------------------------
__global__ void KerInOutInterpolateTime(unsigned npt,double fxtime
  ,const float *vel0,const float *vel1,float *vel)
{
  const unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<npt){
    const float v0=vel0[p];
    vel[p]=float(fxtime*(vel1[p]-v0)+v0);
  }
}

//==============================================================================
/// Interpolate data between time0 and time1.
//==============================================================================
void InOutInterpolateTime(unsigned npt,double time,double t0,double t1
  ,const float *velx0,const float *velx1,float *velx
  ,const float *velz0,const float *velz1,float *velz)
{
  if(npt){
    const double fxtime=((time-t0)/(t1-t0));
    dim3 sgrid=cusph::GetGridSize(npt,SPHBSIZE);
    KerInOutInterpolateTime <<<sgrid,SPHBSIZE>>> (npt,fxtime,velx0,velx1,velx);
    if(velz0)KerInOutInterpolateTime <<<sgrid,SPHBSIZE>>> (npt,fxtime,velz0,velz1,velz);
  }
}

//------------------------------------------------------------------------------
/// Interpolate velocity in time and Z-position of selected partiles in a list.
//------------------------------------------------------------------------------
__global__ void KerInOutInterpolateZVel(unsigned izone,double posminz,double dpz,int nz1
  ,const float *velx,const float *velz
  ,unsigned np,const int *plist,const double *posz,const typecode *code,float4 *velrhop)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<np){
    const unsigned p=plist[cp];
    if(izone==CODE_GetIzoneFluidInout(code[p])){
      const double pz=posz[p]-posminz;
      int cz=int(pz/dpz);
      cz=max(cz,0);
      cz=min(cz,nz1);
      const double fz=(pz/dpz-cz);  //const double fz=(pz-Dpz*cz)/Dpz;
      //-Interpolation in Z.
      const unsigned cp=cz;
      const float v00=velx[cp];
      const float v01=(cz<nz1? velx[cp+1]: v00);
      const float v=float(fz*(v01-v00)+v00);
      velrhop[p]=make_float4(v,0,0,velrhop[p].w);
      if(velz!=NULL){
        const float v00=velz[cp];
        const float v01=(cz<nz1? velz[cp+1]:    v00);
        const float v=float(fz*(v01-v00)+v00);
        velrhop[p].z=v;
      }
    }
  }
}

//==============================================================================
/// Interpolate velocity in time and Z-position of selected partiles in a list.
//==============================================================================
void InOutInterpolateZVel(unsigned izone,double posminz,double dpz,int nz1
  ,const float *velx,const float *velz
  ,unsigned np,const int *plist,const double *posz,const typecode *code,float4 *velrhop)
{
  if(np){
    dim3 sgrid=cusph::GetGridSize(np,SPHBSIZE);
    KerInOutInterpolateZVel <<<sgrid,SPHBSIZE>>> (izone,posminz,dpz,nz1,velx,velz,np,plist,posz,code,velrhop);
  }
}

//------------------------------------------------------------------------------
/// Removes interpolated Z velocity of inlet/outlet particles.
//------------------------------------------------------------------------------
__global__ void KerInOutInterpolateResetZVel(unsigned izone,unsigned np,const int *plist
  ,const typecode *code,float4 *velrhop)
{
  const unsigned cp=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<np){
    const unsigned p=plist[cp];
    if(izone==CODE_GetIzoneFluidInout(code[p]))velrhop[p].z=0;
  }
}

//==============================================================================
/// Removes interpolated Z velocity of inlet/outlet particles.
//==============================================================================
void InOutInterpolateResetZVel(unsigned izone,unsigned np,const int *plist
  ,const typecode *code,float4 *velrhop)
{
  if(np){
    dim3 sgrid=cusph::GetGridSize(np,SPHBSIZE);
    KerInOutInterpolateResetZVel <<<sgrid,SPHBSIZE>>> (izone,np,plist,code,velrhop);
  }
}



}


